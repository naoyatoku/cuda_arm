#include "hip/hip_runtime.h"
#include "SFML/Graphics.hpp"
#include "units.h"
#include "arm.h"	//
#include "CommonModule.h"

HANDLE _draw_req_event; // イベントオブジェクトのハンドル

const int WINDOW_WIDTH = 1500;
const int WINDOW_HEIGHT = 1000;

//中心
const int ORG_X = WINDOW_WIDTH / 2;
const int ORG_Y = WINDOW_HEIGHT / 2;

//このモジュール独自にarmオブジェクトを持ちます。
static arm  __arm;

//アーム状態を

//外部からアームを受けて描画更新をしてみる。
void	draw(const arm& _arm) {
	__arm = _arm;
	::SetEvent(_draw_req_event);
}




//void draw()
DWORD WINAPI draw_thread(LPVOID param) 
{
	_draw_req_event = CreateEvent(NULL, FALSE, FALSE, NULL);
	_Assert(_draw_req_event != NULL, "dreaw_thread : event create failed");

        // ウィンドウの作成
    sf::RenderWindow window(sf::VideoMode(WINDOW_WIDTH, WINDOW_HEIGHT), "arm");
//	window.clear(sf::Color::White);
//	window.display();
	//=====================================
	//		座標軸です。
	//=====================================
	sf::Vector2f y1(ORG_X, 0);	sf::Vector2f y2(ORG_X, ORG_Y * 2);
	sf::Vector2f x1(0, ORG_Y);	sf::Vector2f x2(ORG_X * 2, ORG_Y);
	sf::VertexArray	axis_y(sf::Lines, 2);
	axis_y[0].position = y1;			axis_y[1].position = y2;
	axis_y[0].color = sf::Color::Black;	axis_y[1].color = sf::Color::Black;
	sf::VertexArray	axis_x(sf::Lines, 2);
	axis_x[0].position = x1; 			axis_x[1].position = x2;
	axis_x[0].color = sf::Color::Black;	axis_x[1].color = sf::Color::Black;

	//line?これは何のためかわからない。
/*	sf::Vector2f p1(100, 100);
	sf::Vector2f p2(700, 500);
	sf::VertexArray line(sf::Lines, 2);
	line[0].position = p1;
	line[1].position = p2;
*/
	window.clear(sf::Color::White);
	//
	sf::RectangleShape l1(sf::Vector2f(230.f, 3.f));    l1.setFillColor(sf::Color::Blue);
	sf::RectangleShape l2(sf::Vector2f(210.f, 2.f));    l2.setFillColor(sf::Color::Green);
	sf::RectangleShape l3(sf::Vector2f(144.f, 2.f));    l3.setFillColor(sf::Color::Magenta);


	_cood before;
	while (window.isOpen()) {
		// イベント処理
		sf::Event event;
		while (window.pollEvent(event)) {
			if (event.type == sf::Event::Closed)
				window.close();
		}

		//同期です
		{
			// イベントを待機
			WaitForSingleObject(_draw_req_event, 1000);
		}
//		if (__arm.operator==(before)) {
//			continue;
//		}
//		printf("D");

		window.clear(sf::Color::White);

		//各アームを動作させる
		l1.setPosition(ORG_X, ORG_Y);				//これはリンクの先端の座標。
		l1.setRotation(-1 * _deg(__arm.lnk(0).r));	//角度をそこにする。y軸は反対になる。
		//この場合、根本の座標なんで一つ前の軸の先端座標を指定しないといけない。
		l2.setPosition(ORG_X + __arm.lnk(0).linked_vect().x, ORG_Y - __arm.lnk(0).linked_vect().y);	//l1の先端座標にする。
		l2.setRotation(-1 * _deg(__arm.lnk(0).r + __arm.lnk(1).r));								//角度をそこにする。
		l3.setPosition(ORG_X + __arm.lnk(1).linked_vect().x, ORG_Y - __arm.lnk(1).linked_vect().y);	//21の先端座標にする。
		l3.setRotation(-1 * _deg(__arm.lnk(0).r + __arm.lnk(1).r + __arm.lnk(2).r));	//角度をそこにする。

		//
		// 
		
		//=============
		//	draw
		//=============
		window.draw(l1);				//
		window.draw(l2);				//
		window.draw(l3);				//
		//座標軸です。
		window.draw(axis_x);	window.draw(axis_y);

		window.display();


		before = __arm;
//		::Sleep(1);
	}
	return 0;
}