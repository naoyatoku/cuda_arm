#include "hip/hip_runtime.h"
#include "SFML/Graphics.hpp"
#include "units.h"
#include "arm.h"	//
#include "CommonModule.h"

HANDLE _draw_req_event; // イベントオブジェクトのハンドル

const int WINDOW_WIDTH = 1500;
const int WINDOW_HEIGHT = 1000;

//中心
const int ORG_X = WINDOW_WIDTH / 2;
const int ORG_Y = WINDOW_HEIGHT / 2;


//void draw()
DWORD WINAPI draw_thread(LPVOID param) 
{
	_draw_req_event = CreateEvent(NULL, TRUE, FALSE, NULL);
	_Assert(_draw_req_event != NULL, "dreaw_thread : event create failed");

        // ウィンドウの作成
    sf::RenderWindow window(sf::VideoMode(WINDOW_WIDTH, WINDOW_HEIGHT), "arm");
//	window.clear(sf::Color::White);
//	window.display();
	//=====================================
	//		座標軸です。
	//=====================================
	sf::Vector2f y1(ORG_X, 0);	sf::Vector2f y2(ORG_X, ORG_Y * 2);
	sf::Vector2f x1(0, ORG_Y);	sf::Vector2f x2(ORG_X * 2, ORG_Y);
	sf::VertexArray	axis_y(sf::Lines, 2);
	axis_y[0].position = y1;			axis_y[1].position = y2;
	axis_y[0].color = sf::Color::Black;	axis_y[1].color = sf::Color::Black;
	sf::VertexArray	axis_x(sf::Lines, 2);
	axis_x[0].position = x1; 			axis_x[1].position = x2;
	axis_x[0].color = sf::Color::Black;	axis_x[1].color = sf::Color::Black;

	//line?これは何のためかわからない。
/*	sf::Vector2f p1(100, 100);
	sf::Vector2f p2(700, 500);
	sf::VertexArray line(sf::Lines, 2);
	line[0].position = p1;
	line[1].position = p2;
*/
	window.clear(sf::Color::White);
	//
	sf::RectangleShape l1(sf::Vector2f(230.f, 3.f));    l1.setFillColor(sf::Color::Blue);
	sf::RectangleShape l2(sf::Vector2f(210.f, 2.f));    l2.setFillColor(sf::Color::Green);
	sf::RectangleShape l3(sf::Vector2f(144.f, 2.f));    l3.setFillColor(sf::Color::Magenta);

	while (window.isOpen()) {
		// イベント処理
		sf::Event event;
		while (window.pollEvent(event)) {
			if (event.type == sf::Event::Closed)
				window.close();
		}

		//同期です
		{
			// イベントを待機
			WaitForSingleObject(_draw_req_event, 1000);
		}

		window.clear(sf::Color::White);

		//各アームを動作させる
		l1.setPosition(ORG_X, ORG_Y);				//これはリンクの先端の座標。
		l1.setRotation(-1 * _deg(_arm.lnk(0).r));	//角度をそこにする。y軸は反対になる。
		//この場合、根本の座標なんで一つ前の軸の先端座標を指定しないといけない。
		l2.setPosition(ORG_X + _arm.lnk(0).linked_vect().x, ORG_Y - _arm.lnk(0).linked_vect().y);	//l1の先端座標にする。
		l2.setRotation(-1 * _deg(_arm.lnk(0).r + _arm.lnk(1).r));								//角度をそこにする。
		l3.setPosition(ORG_X + _arm.lnk(1).linked_vect().x, ORG_Y - _arm.lnk(1).linked_vect().y);	//21の先端座標にする。
		l3.setRotation(-1 * _deg(_arm.lnk(0).r + _arm.lnk(1).r + _arm.lnk(2).r));	//角度をそこにする。

		//
		// 
		
		//=============
		//	draw
		//=============
		window.draw(l1);				//
		window.draw(l2);				//
		window.draw(l3);				//
		//座標軸です。
		window.draw(axis_x);	window.draw(axis_y);

		window.display();

//		::Sleep(1);
	}
	return 0;
}