﻿#include "hip/hip_runtime.h"
#include ""
#include "commonmodule.h"
#include "path.h"
//=============================================================================================
//      軌跡データ
//=============================================================================================
/*


double _path_data[_PATH_DATA_MAX];
int _path_idx;


void init_path()
{
    memset((void*)&_path_data[0] , 0 , sizeof(_path_data) );
    _path_idx = 0;
}
//GPUから軌跡データへ追記します。
void append_path(void* gpu_src, void* gpu_n,int offset)
{
    hipError_t s;
    int n;
    //データ数の取得です。
    s = hipMemcpy((void*)&n, gpu_n, sizeof(int), hipMemcpyDeviceToHost);
    _Assert(s == hipSuccess, "hipMalloc failed!" );
    //データ本体の取り込みです。
    _Assert(_path_idx + n < _PATH_DATA_MAX , "path data overflow\n" );
    s = hipMemcpy((void*)&_path_data[_path_idx+offset] , gpu_src ,  n * sizeof(double), hipMemcpyDeviceToHost );
    _Assert(s == hipSuccess, "hipMalloc failed!");
    //インデックス更新です。
    _path_idx += n;
}
*/