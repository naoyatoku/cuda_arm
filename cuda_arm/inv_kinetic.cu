#include "hip/hip_runtime.h"
#include	"inv_kinetic.h"
#include	<math.h>
#include	<stdio.h>
//------------------------------------------------------------------------------------------------------------
//		defineq
//------------------------------------------------------------------------------------------------------------


//------------------------------------------------------------------------------------------------------------
//		global
//------------------------------------------------------------------------------------------------------------
//�v�Z�̓s����Adouble�ŃA�[�������������܂��B
//�ȉ��̓f�t�H���g�ł��B�v��Ȃ�����

//__host__ __device__ void set_kinetic_parameter(double l1,double l2 , double l3);


//���Ƃ��Ă����ɂ����Ă����܂��B

/*
struct _3link_calc
{
	float _l1, _l2, _l3;

	__device__ __host__ float	_a(float y, float rad)
	{
		float a = y - (_l3 * sin(rad));
		return a;
	}
	__device__ __host__ float	_b(float x, float rad)
	{
		float	b = x - (_l3 * cos(rad));
		return b;
	}
	__device__ __host__ float	_c(float x, float y, float rad)
	{
		float a = _a(y, rad);
		float b = _b(x, rad);
		float c = ((a * a) + (b * b) + (_l1 * _l1) - (_l2 * _l2)) / (2 * _l1);
		return c;
	}
	__device__ __host__ float		_d(float x, float y, float rad)
	{
		float a = _a(y, rad);
		float b = _b(x, rad);
		float d = ((a * a) + (b * b) - (_l1 * _l1) + (_l2 * _l2)) / (2 * _l1);
		return d;
	}
public:
	//���̃��W���[���Ŏg���Ԑڂ̃A�[��������o�^�o����悤�ɂ��Ă����܂��B
	__device__ __host__ void set_kinetic_parameter(float l1, float l2, float l3)
	{
		_l1 = l1;
		_l2 = l2;
		_l3 = l3;
	}


		//3�v�f�̔z��F 

	//	__device__ __host__ vector<Vector3d> calc_inv_kinetic(double x, double y, double rad)
	__device__ __host__ mlti< _Vector3d<float>, 2>  calc_inv_kinetic(float x, float y, float rad)
	{
		//	vector<Vector3d> r(2);
		mlti< _Vector3d<float>, 2> _r;

		//	Vector3d	r[2];			//������ʂ�A�֐߂�3�ł��B
		float a = _a(y, rad);
		float b = _b(x, rad);
		float c = _c(x, y, rad);
		float d = _d(x, y, rad);
		float phy = atan2(a, b);
		//		printf("a:%lf , b:%lf , c:%lf , d:%lf , phy:%lf ,\r\n", a, b, c, d, phy);
#if 1
		//---------------------------
		//	��1
		//---------------------------
		float	_sub_1 = atan2(sqrt((a * a) + (b * b) - (c * c)), c);
		//		r[0](0) = phy - _sub_1;
		//		r[1](0) = phy + _sub_1;		//����͕K�v�Ȃ���Ώ����B
				//{for(int i=0;i<2;++i){	printf("q1[%d]  = %lf \r\n"  , i , r.q1[i] );}}
		_r[0](0) = phy - _sub_1;
		_r[1](0) = phy + _sub_1;
		//---------------------------
		//	��2
		//---------------------------
		float _sub_2 = atan2(sqrt((a * a) + (b * b) - (d * d)), d);
		//		r[0](1) = phy - r[0][0] + _sub_2;
		//		r[1](1) = phy - r[1][0] - _sub_2;
		_r[0](1) = phy - _r[0][0] + _sub_2;
		_r[1](1) = phy - _r[1][0] - _sub_2;

		//{for(int i=0;i<2;++i){	printf("q2[%d]  = %lf \r\n"  , i , r.q2[i] );}}
			//---------------------------
			//	��3
			//---------------------------
//		r[0](2) = rad - (r[0][0] + r[0][1]);
//		r[1](2) = rad - (r[1][0] + r[1][1]);
		_r[0](2) = rad - (_r[0][0] + _r[0][1]);
		_r[1](2) = rad - (_r[1][0] + _r[1][1]);

		//�Ƃ肠�����A2�΂̒��ɐ�����[�߂܂��B
		for (int k = 0; k < 2; ++k) {
			for (int i = 0; i < 3; ++i) {
				_r[k](i) = fmodf(_r[k](i), (2.0 * PI));		//�܂�2�΂̒��ɔ[�߂܂��B
				//				_gpuAssert( ! std::isnan(_r[k](i)) , "calc_inv_kinetick is nan" );
			}
		}

		//printf("q3 = %lf \r\n" , r.q3[0] );cd
		//{for(int i=0;i<2;++i){	printf("q3[%d]  = %lf \r\n"  , i , r.q3[i] );}}
		//	_dump_theta(r);
#endif
		return _r;
	}

	__device__ __host__  mlti< _Vector3d<float>, 2> calc_inv_kinetic(_cood p) {
		// calc_inv_kinetic(_cood p){
		return calc_inv_kinetic(p.x, p.y, p.rad);
	}
	//------------------------------------------------------------------------------------------------------------
	//	3 link ���ʃA�[����  ���^���w
	//-----------------------------------------------------------------------------------------------------------
	__device__ __host__ _cood calc_fwd_kinetic(float rad[3])
	{
		//3�̃A�[���̃x�N�g�������Z�ł��B
		_cood p;
		p.x = (_l1 * cos(rad[0])) + (_l2 * cos(rad[0] + rad[1])) + (_l3 * cos(rad[0] + rad[1] + rad[2]));
		p.y = (_l1 * sin(rad[0])) + (_l2 * sin(rad[0] + rad[1])) + (_l3 * sin(rad[0] + rad[1] + rad[2]));
		p.rad = rad[0] + rad[1] + rad[2];
		return p;
	}

	//���R�r��������
	//toku �Ƃ肠����cuda�ł�Eigen���C�u�������g���Ȃ��̂ŁA���R�r�֘A�͕ۗ��A
	__host__  Matrix3d jacobian(float th1, float th2, float th3)
	{
		Eigen::Matrix3d J;

		J << -_l1 * sin(th1) - _l2 * sin(th1 + th2) - _l3 * sin(th1 + th2 + th3), -_l2 * sin(th1 + th2) - _l3 * sin(th1 + th2 + th3), -_l3 * sin(th1 + th2 + th3),
			_l1* cos(th1) + _l2 * cos(th1 + th2) + _l3 * cos(th1 + th2 + th3), _l2* cos(th1 + th2) + _l3 * cos(th1 + th2 + th3), _l3* cos(th1 + th2 + th3),
			1, 1, 1;
		return J;
	}

	__host__ bool inv_jacobi(float th1, float th2, float th3, Matrix3d& J_inv)
	{
		// Calculate the Jacobian
		Matrix3d J = jacobian(th1, th2, th3);

		// Check if the determinant is non-zero
		{
			float det = J.determinant();
			//			if (_gpuAssert(std::abs(det) > 1e-6, "The determinant is zero. The inverse Jacobian does not exist.") != true) {				return false;			}
			if (std::abs(det) < 1e-6) {
				printf("The determinant is zero. The inverse Jacobian does not exist.");
				return false;
			}
		}

		// Calculate the inverse Jacobian
		J_inv = J.inverse();
		return true;
	}
};		//class 
*/