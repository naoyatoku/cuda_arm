#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "CommonModule.h"
#include "draw.h"
#include "circle.h"
#include "arm.h"
#include "path.h"

#define N_THREAD        512         //一ブロックで実行するスレッドの数はdefineでcudaカーネルでも使えるようにします。
#define N_BLOCK         512

#define GPU_PATH_MAX    2048        //GPU側に送るパスのバッファサイズです。
#define PATH_MAX        4096        //CPU側でまとめて保存するバッファサイズです。_path_data[]本体の

//円形に動作する
// 一定速動作、
//減速停止

//hipMallocは、グローバルメモリに保存される
//まず大きなグローバルメモリをとり、一回分の計算を格納する。
//違うカーネルで全ブロック分の最大の分布をみて、経路を完成させる。

//現在の位置から _arm 
__global__ void circle_calc_kernel(arm* _arm , circle* _c ,int* succeed , int *succeed_blc ,int dir,double init_spd ,  float spd )
{
    //初速と目標速度との関係で減速するのか、加速するのかを決めます。
    __shared__ bool             acc;                    //加速するかどうか
    __shared__ double           spd_span;               //速度の幅
    __shared__ int              default_idx;            //
    __shared__ int     reduction[N_THREAD];     //

    //ブロックとスレッドの組み合わせで成功失敗の
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    //__shared__メモリの初期化です.各ブロックに
    if (threadIdx.x == 0) {
        acc = (spd > init_spd) ? true : false;    //加減速スイッチです。
        spd_span = fabs(spd - init_spd);          //加減速による速度の幅です。（初速度～目標速度間の幅）
        default_idx = acc ? -1 : (blockDim.x*gridDim.x);        //（成功しない場合に
    }
    __syncthreads();        //あるブロック内部のスレッドの同期です。
    //今回チャレンジする移動距離をdtとします。
    double dt = dir * (spd_span / (N_BLOCK*N_THREAD)) * (idx + (acc ? 1 : 0)) / 1000; //1ms分なので、 spd rad/sec -> rad/msec に修正        idx=511で bloclDim.xとなるように
    if (blockIdx.x == 511 && threadIdx.x == 511) {          //debug stopeer
        reduction[threadIdx.x] = succeed[idx];               //高速にリダクションするためにsharedに入れます。
    }
    link_stat s = _arm->move_able(_cood(_c->progress_rad(*_arm, dt, dir), _arm->rad + dt));


    succeed[idx] = s.stat == MOV_OK ? idx : default_idx;     //成功したら自分のインデックスを入れる
    reduction[threadIdx.x] = succeed[idx];               //高速にリダクションするためにsharedに入れます。

    //※失敗の場合のデフォルト値は、減速の場合には小さいほうが採用されるように大きな値にしておく。
    // （idx : 0 - blockDim.x-1 なので、 ）
//     __device__ __host__ vec2 progress_rad(const vec2 & start, double rad, int dir = 0) const;
//        printf("[%d]:%d %d\n", idx, result[idx], succeed[idx]);
    __syncthreads();        //あるブロック内部のスレッドの同期です。
    //==================================================================================
    // リダクションでブロック単位の評価を行います。
    //===================================================================================
    {   //リダクション。減速と加速で条件が違います。
        //条件分岐が少ないように重複して書いてみる。本当にパフォーマンス変わるのか
        int _idx = threadIdx.x;     //ここで使われるのはスレッドのインデックスです。
        if (acc) {
            for (int i = blockDim.x / 2; i > 0; i /= 2) {
                if (_idx < i && reduction[_idx + i] > reduction[_idx]) {
                    reduction[_idx] = reduction[_idx + i];    //半分より大きな部分と比較して大きなほうを
                }
                __syncthreads();
            }
        }
        else {
            for (int i = blockDim.x / 2; i > 0; i /= 2) {
                if (_idx < i && reduction[_idx + i] < reduction[_idx]) {       //減速の場合は小さいほうを選択する。
                    reduction[_idx] = reduction[_idx + i];    //半分より大きな部分と比較して大きなほうを
                }
                __syncthreads();
            }
        }
        //ここでreduction[0]が、このブロック内で一番いい値です。
        if (threadIdx.x == 0) { 
            succeed_blc[blockIdx.x] = reduction[0]; 
        }
    }

}
//成功配列から、一番効率のいいものを選択して実際にアームを動作させる
__global__ void circle_move_kernel(arm* _arm, circle* _c, int* succeed, int* succeed_blc,  double* path, int* n_path,int *fin , int dir , double e_rad , double init_spd , double spd,bool pos , int kernelno=0)
{
    __shared__ bool             acc;                    //加速するかどうか
    __shared__ double           spd_span;               //速度の幅
    __shared__ int              default_idx;            //
    //ブロックとスレッドの組み合わせで成功失敗の
#if 1       //debug
    if (kernelno == 20) {       //20番カーネルが、arm状態があっているかどうか
        __syncthreads();
    }
#endif

    int idx = threadIdx.x;
    //__shared__メモリの初期化です.各ブロックに
    if (threadIdx.x == 0) {
        acc = (spd > init_spd) ? true : false;                  //加減速スイッチです。
        spd_span = fabs(spd - init_spd);                        //加減速による速度の幅です。（初速度～目標速度間の幅）
//        default_idx = acc ? -1 : (blockDim.x * gridDim.x);          //（成功しない場合に
        default_idx = acc ? -1 : 0x7fffffff;    //int最大
    }
    __syncthreads();
    //==================================================================================
    // リダクションでブロック単位の評価を行います。
    //===================================================================================
    int adopted_idx;        //これが採用する
    {   //リダクション。減速と加速で条件が違います。
        //条件分岐が少ないように重複して書いてみる。本当にパフォーマンス変わるのか
        int _idx = threadIdx.x;     //ここで使われるのはスレッドのインデックスです。
        if (acc) {
            for (int i = blockDim.x / 2; i > 0; i /= 2) {
                if (_idx < i && succeed_blc[_idx + i] > succeed_blc[_idx]) {
                    succeed_blc[_idx] = succeed_blc[_idx + i];    //半分より大きな部分と比較して大きなほうを
                }
                __syncthreads();
            }
        }
        else {
            for (int i = blockDim.x / 2; i > 0; i /= 2) {
                if (_idx < i && succeed_blc[_idx + i] < succeed_blc[_idx]) {       //減速の場合は小さいほうを選択する。
                    succeed_blc[_idx] = succeed_blc[_idx + i];    //半分より大きな部分と比較して大きなほうを
                }
                __syncthreads();
            }
        }
        //ここでscceed_[0]が、このブロック内で一番いい値です。
        if (threadIdx.x == 0) {
            adopted_idx = succeed_blc[0];   //succeed_blcには、ブロック、スレッドを通したインデックスが入っています。
        }
    }

    //選択されたインデックスでアームを動かし、その結果条件にあっているかを判定します。
    if (threadIdx.x == 0) {
        if (adopted_idx < 0) {  //一つも動かせない場合
            printf("cant move\n");     *fin = -1;       goto error;
        }
//        if (pos && (*n_path > 105)) {            printf("-");        }
        double dt = dir * (spd_span / (N_BLOCK * N_THREAD)) * (adopted_idx + (acc ? 1 : 0)) / 1000;     //1ms分なので、 spd rad/sec -> rad/msec に修正        idx=511で bloclDim.xとなるように
        _arm->move(_cood(_c->progress_rad(*_arm, dt, dir), _arm->rad + dt));                            //動かす。
//        printf("[%d]t:%lf,spd:%lf,acc%lf,dlt:%lf\n", *n_path, _arm->rad, _arm->wd.spd, _arm->wd.acc, fabs(_arm->wd.spd) - spd);
        // 記録します。(dt)
        path[(*n_path)++] = dt;

        //===============================================
        //  終了条件の判定をします。
        //===============================================
        if (pos) {
            //位置の場合には、ピッタリとまらないことがあるので、行きすぎたら終わる。
//              //もしくは、次の加速で
            if ((dir == CIRCLE_DIR_CW && _arm->rad > e_rad)
                || (dir == CIRCLE_DIR_CCW && _arm->rad < e_rad)) {
                printf("pos toutatu!!!\n");
                *fin = (int)true;
            }
            else {
                *fin = (int)false;
            }
        }
        else {
            //動かしたところで、目標到達速度に達すると終了。目標速度
            if (_equal( (double)fabs(_arm->wd.spd), (double)spd,0.0005)) {   //精度はちょっと甘くしてみます。
                printf("spd toutatu!!! (%d)-(%d)(%d)\n" , kernelno,blockIdx.x,threadIdx.x);
                *fin = (int)true;
            }
            else {
                *fin = (int)false;
            }
        }
    }
    return;
error:
    //finに結果が表示されます。
    return;
}



#define N_SEARCH    32     //着地探索計算の回数。

//
void circle_path_2()
{
//    const int threads   = 512;    //
//    const int blocks    =   128;    //

    circle* p_c;
    arm* p_arm;

    //計算分を格納するバッファを作ります。
    static  int    succeed[N_THREAD * N_BLOCK];       //staticとしてみる。
    static  int    succeed_blc[N_BLOCK];       //staticとしてみる。
    int *p_succeed;                                //デバイス側です。
    int* p_succeed_blc;                          //ブロック単位でのリダクション結果を格納するための

    //経路データです。
    double* p_path;     //GPUに渡す軌跡データ
    int* p_n_path;     //GPUで計算した軌跡の数
    hipError_t s;

    //アームが目的の状態になったかどうか（カーネルが判断した結果を格納します。)
    //int fin;
    int* p_fin;

    //==========================================================================
    //  軌跡となる円を設定します。
    //==========================================================================
    circle c(100, 50, 50);


    //==========================================================================
    init_path();                    //軌跡の記録データのセットアップです。
    //==========================================================================


    //試しに、0から90度まで、初速0 - 最高速 90/sec- 終了速度 0 で動ける軌跡を探索する。
    //======= 条件です ==============================
    double s_rad = 0;               //姿勢も角度と同じとしてみます。
    double e_rad = PI / 2;            //b
    double spd = PI / 2;              //PI/2の速度まで
    //======= 条件です ==============================
    //armサークルオブジェクトをgpuへ転送
    {
        //
        s = hipMalloc((void**)&p_c, sizeof(circle));                               _Assert(s == hipSuccess, "hipMalloc failed!");
        s = hipMemcpy(p_c, (void*)&c, sizeof(circle), hipMemcpyHostToDevice);     _Assert(s == hipSuccess, "hipMalloc failed!");        //
        s = hipMalloc((void**)&p_arm, sizeof(arm));                                _Assert(s == hipSuccess, "hipMalloc failed!");        //
        //失敗成功(全データを見られるように)    ※
        s = hipMalloc((void**)&p_succeed       , sizeof(succeed));                  _Assert(s == hipSuccess, "hipMalloc failed!");
        s = hipMalloc((void**)&p_succeed_blc   , sizeof(succeed_blc));              _Assert(s == hipSuccess, "hipMalloc failed!");
        //ブロック単位での成功失敗を格納するためのメモリです。
//        s = hipMalloc()
        //経路の記録。
        s = hipMalloc((void**)&p_path, sizeof(double) * PATH_MAX);                  _Assert(s == hipSuccess, "hipMalloc failed!");        //軌跡データ
        s = hipMalloc((void**)&p_n_path, sizeof(int));                              _Assert(s == hipSuccess, "hipMalloc failed!");        //軌跡データの数
        //カーネルの判定用。
        s = hipMalloc((void**)&p_fin, sizeof(int));                                 _Assert(s == hipSuccess, "hipMalloc failed!");        //軌跡データの数
    }

    //==========================================================================================================
    //   到達位置から逆に加速させ、目標速度に到達するようにする。
    //==========================================================================================================
    double dec_start_pos;       //減速開始位置です。
    {
        _arm.set(_cood(c.rf(e_rad), e_rad));           //アームを終了位置へセットします。
        s = hipMemcpy(p_arm, (void*)&_arm, sizeof(arm), hipMemcpyHostToDevice);   _Assert(s == hipSuccess, "hipMalloc failed!");

        for (int fin = 0 ; !fin ;) {
            //一回分のカーネルです。
            {
                circle_calc_kernel << <N_BLOCK, N_THREAD >> > (p_arm, p_c, p_succeed, p_succeed_blc, CIRCLE_DIR_CCW, 0 , spd);
                hipDeviceSynchronize();       s = hipGetLastError();        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
            }
#if 1   //このカーネルではp_succeedに

#endif
            //ここで各ブロックの集計結果をまとめて、一番いいものを選択し、実際にアームを動作させ、軌跡を記録します。
            {
                circle_move_kernel << <1, N_BLOCK >> > (p_arm, p_c, p_succeed, p_succeed_blc, p_path, p_n_path, p_fin, CIRCLE_DIR_CCW , e_rad , 0 ,spd ,  false);        //速度が到達したらおしまい。
                hipDeviceSynchronize();        //終了待ち
            }
            //結果をCPUに転送して評価していきます。ここが一番時間がかかる。
            { s = hipMemcpy((void*)&fin, (void*)p_fin, sizeof(int), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
        }
        printf("save path data..");
        //    append_path(p_path, p_n_path);  //軌跡データです。
        //現時点のアームの位置が、おおよその減速開始位置です。
        s = hipMemcpy((void*)&_arm, p_arm, sizeof(arm), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMalloc failed!");

        //
        dec_start_pos = _arm.rad;
        printf("fin\n");
    }
    //==========================================================================================================
    //  初期位置から減速位置まで動作させていく。
    //  探索のために、ちょっと行きすぎるようにしてみます。
    //==========================================================================================================
    init_path();        //軌跡はクリアです。

    //
    {   //GPU側の軌跡カウンタクリアと、アームの初期位置のセットです。
        int n_path = 0;
        s = hipMemcpy(p_n_path, (void*)&n_path, sizeof(int), hipMemcpyHostToDevice);     _Assert(s == hipSuccess, "hipMalloc failed!");        //
        _arm.set(_cood(c.rf(s_rad), s_rad));           //アームを開始位置へセットします。
        s = hipMemcpy(p_arm, (void*)&_arm, sizeof(arm), hipMemcpyHostToDevice);   _Assert(s == hipSuccess, "hipMalloc failed!");
    }
    printf("start search path...");
    {   //探索
        int i = 0;
//        double ov = fabs( e_pos - dec_start_pos)     ;        //少し行きすぎる場所まで記録する。
        for (int fin = 0; !fin; ++i) {
            //一回分のカーネルです。
            {
                circle_calc_kernel << <N_BLOCK, N_THREAD >> > (p_arm, p_c, p_succeed, p_succeed_blc, CIRCLE_DIR_CW, 0 , spd);
                hipDeviceSynchronize();    s = hipGetLastError();        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
            }
/*            if (i>105) {
                { s = hipMemcpy((void*)&succeed[0], (void*)p_succeed, sizeof(succeed), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
                { s = hipMemcpy((void*)&succeed_blc[0], (void*)p_succeed_blc, sizeof(succeed_blc), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
            }
            */
            //ここで各ブロックの集計結果をまとめて、一番いいものを選択し、実際にアームを動作させ、軌跡を記録します。
            {
                //※ちょっと行きすぎる場所まで
                circle_move_kernel << <1, N_BLOCK >> > (p_arm, p_c, p_succeed, p_succeed_blc, p_path, p_n_path, p_fin, CIRCLE_DIR_CW, dec_start_pos + _rad(1), 0, spd, true);        //位置を目標に
                hipDeviceSynchronize();        //終了待ち
            }
            //結果をCPUに転送して評価していきます。ここが一番時間がかかる。
            { s = hipMemcpy((void*)&fin, (void*)p_fin, sizeof(int), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
        }
        printf("fin(loop : %d)\n", i);
    }
    //軌跡をアップロードします。
    append_path(p_path, p_n_path);  //これでコピーします
    //==========================================================================================================
    //  ここで、_path_data[_n_path] には０～radまでの軌跡が記録されている。
    //  後半のいくつかを使って、そこから全力で減速させてみる。
    //  もっとも目的位置に近い場所て停止した軌跡を採用するようにする。
    //==========================================================================================================
    {
        //
        static arm _s_arm[N_SEARCH];                 //探索開始位置の

        //一応探索結果も
        static  int    s_succeed[N_SEARCH*N_THREAD * N_BLOCK];       //staticとしてみる。
        static  int    s_succeed_blc[N_SEARCH*N_BLOCK];       //staticとしてみる。
        arm* p_s_arm;          //

        int* p_s_succeed;                                //デバイス側です。
        int* p_s_succeed_blc;                          //ブロック単位でのリダクション結果を格納するための

        double* p_s_path;       //GPUに渡す軌跡データ
        int* p_n_s_path;        //GPUで計算した軌跡の数

        //各軌跡計算の終了状況。
        int* p_s_fin;
        int s_fin[N_SEARCH];

        //=============================================================================
        //      アーム、経路データは探索回数分用意します。メモリ
        //      
        //=============================================================================
        {
            s = hipMalloc((void**)&p_s_arm, sizeof(_s_arm));                                _Assert(s == hipSuccess, "hipMalloc failed!");        //

            //GPUで記録される軌跡データです。
            s = hipMalloc((void**)&p_s_path, sizeof(double) * PATH_MAX * N_SEARCH);            _Assert(s == hipSuccess, "hipMalloc failed!");
            s = hipMalloc((void**)&p_n_s_path, sizeof(int) * N_SEARCH);                          _Assert(s == hipSuccess, "hipMalloc failed!");        //
            //
            //失敗成功(全データを見られるように)    ※
            s = hipMalloc((void**)&p_s_succeed, sizeof(s_succeed));                                  _Assert(s == hipSuccess, "hipMalloc failed!");
            s = hipMalloc((void**)&p_s_succeed_blc, sizeof(s_succeed_blc));                          _Assert(s == hipSuccess, "hipMalloc failed!");

            //終了
            s = hipMalloc((void**)&p_s_fin, sizeof(int) * N_SEARCH);                                 _Assert(s == hipSuccess, "hipMalloc failed!");        //軌跡データの数
        }

        //軌跡一つは1ms分のデータです。最後の128msくらいまでを試します。s
        int st = max( _path_idx - N_SEARCH , 0);
        _arm.set(_cood(c.rf(s_rad), s_rad));                                                                                                                  //アームをスタート位置へ移動
        for (int i = 0; i < st; ++i) {  _arm.move(_cood(c.progress_rad(_arm, _path_data[i], CIRCLE_DIR_CW), _arm.rad + _path_data[i]));     }                 //
        //ここから終点まではアーム状態を記録していきます。

        for (int i =st ; i < _path_idx ; ++i) {       //
            _arm.move(_cood(c.progress_rad(_arm, _path_data[i], CIRCLE_DIR_CW), _arm.rad + _path_data[i]));     //アームを動かし、
            _s_arm[i-st] = _arm;      //探索開始位置として記録する。
        }

        //ここで _s_arm[] に 減速開始位置までの  「N_SEARCH ms」分のアームが保持されている。
        //N_SEARCH分のarm状態から減速停止していき、それらを


        //gpuに転送します。
        {
            s = hipMemcpy(p_s_arm, (void*)&_s_arm, sizeof(_s_arm), hipMemcpyHostToDevice);   _Assert(s == hipSuccess, "hipMalloc failed!");
        }
        //============================================================================================
        //  ここから探索していく。探索分カーネルを起動できればうれしいが
        //============================================================================================
        for (int i = 0; i < N_SEARCH; ++i) s_fin[i] = false;     //fin初期化です。
        for (int i=0; ; ++i) {
            printf("[%d]",i);


            //速度０になるまで、おのおののアーム状態（s_arm[t] : gpu:p_s_arm[t]）を動作させるカーネルを
            {
                for (int t = 0; t < N_SEARCH; ++t) {        //複数個カーネルを起動します。
                        //すでに終了しているカーネルは起動しない
                    if (s_fin[t])continue;
                        circle_calc_kernel << <N_BLOCK, N_THREAD >> > (  &p_s_arm[t]
                            , p_c
                            , &p_s_succeed[t * N_THREAD * N_BLOCK]
                            , &p_s_succeed_blc[t * N_BLOCK]
                            , CIRCLE_DIR_CW
                            , _s_arm[t].wd.spd ,0);
                }
                hipDeviceSynchronize();    s = hipGetLastError();        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
            }
#if 1
            if (1) {
               { s = hipMemcpy((void*)&s_succeed[0]        ,   (void*)p_s_succeed      ,   sizeof(s_succeed)       , hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
               { s = hipMemcpy((void*)&s_succeed_blc[0]    ,   (void*)p_s_succeed_blc  ,   sizeof(s_succeed_blc)   , hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }                
            }
#endif

            //これがmoveカーネル。サーチ分動作させる。
            //ここで各ブロックの集計結果をまとめて、一番いいものを選択し、実際にアームを動作させ、軌跡を記録します。
            {
                //※ちょっと行きすぎる場所まで
                for (int t = 0; t < N_SEARCH; ++t) {
                    if (s_fin[t])continue;
                    circle_move_kernel << <1, N_BLOCK >> > (
                          &p_s_arm[t] 
                        , p_c
                        , &p_s_succeed[t * N_THREAD * N_BLOCK]
                        , &p_s_succeed_blc[t * N_BLOCK] 
                        , &p_s_path[ t * PATH_MAX]
                        , &p_n_s_path[ t ]
                        , &p_s_fin[t]
                        , CIRCLE_DIR_CW, 0, _s_arm[t].wd.spd, 0, false , t);        //速度が０になるのを
                }
                hipDeviceSynchronize();    s = hipGetLastError();        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
            }
            //結果をCPUに転送して評価していきます。ここが一番時間がかかる。
            { s = hipMemcpy((void*)&s_fin[0], (void*)p_s_fin, sizeof(int)*N_SEARCH, hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
            //終わっていないものがなければ終了します。
            for (int i = 0; i < N_SEARCH; ++i) {       if(!s_fin[i] ) goto _loop;        }
            break;      //何も実行するものがなくなった。
        _loop:;
        }
        //--------------------------------------------------
        //      ここで終了です。一番目標位置に近い場所を探します。
        //--------------------------------------------------
        int min_idx;
        {
            //止まっているアーム状態を取り出します。
            { s = hipMemcpy((void*)&_s_arm[0], (void*)p_s_arm, sizeof(_s_arm)  , hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
            //
            double min = 10;
            for (int i = 0; i < N_SEARCH; ++i) {
                if (fabs(_s_arm[i].rad - e_rad) < min) {
                    min = fabs(_s_arm[i].rad - e_rad);
                    min_idx = i;
                }
            }
              
        }
        //ここでmin_idxが最強のパスでしたので、それを足していきます。
//        このままじゃだめです。min_idxの場所のパスから足さないといけない。
//       差し込む場所は、、「データ末尾 - N_SEARCH + min_idxの場所」
        append_path(&p_s_path[min_idx* PATH_MAX], &p_n_s_path[min_idx] , min_idx - N_SEARCH+1);



        //メモリ解放を・・・
        {
            hipFree((void*)p_s_arm);
            hipFree((void*)p_s_succeed);                               //デバイス側です。
            hipFree((void*)p_s_succeed_blc);                           //ブロック単位でのリダクション結果を格納するための
            hipFree((void*)p_s_path);                                  //GPUに渡す軌跡データ
            hipFree((void*)p_n_s_path);                                //GPUで計算した軌跡の数
            hipFree((void*) p_s_fin);                                  //
        }

    }

    //
    {
        printf("free gpu memory..");
        hipFree( (void*)p_c);
        hipFree( (void*)p_arm);
        hipFree( (void*) p_succeed);
        hipFree( (void*)p_succeed_blc);
        hipFree( (void*)p_path);
        hipFree( (void*)p_n_path);
        hipFree((void*)p_fin);
        printf("fin\n");
    }

    //ここで計算した軌跡にそって動作させてみます。
    
    {
        printf("arm move..");
        _arm.set(_cood(c.rf(s_rad), s_rad));           //アームをスタート位置へ移動。
        for (int i = 0; i < _path_idx; ++i) {
            if (i > 962) {
                ::Sleep(1);
            }
            _arm.move(_cood(c.progress_rad(_arm, _path_data[i], CIRCLE_DIR_CW), _arm.rad + _path_data[i]));
            //各関節の指示データを保存していきますか
            _arm.save();        //ファイルに保存してみます。
            ::Sleep(1);

        }
        printf("fin\n");
    }

    printf("all fin\n");
    return;
Error:;
    printf("failed");

}



//======================================================================================================================================== 
//  最初の試み   スレッド数だけで分割するので512分割するパターン。ブロックの中で実行できるスレッドが512で制限されてしまった。
//========================================================================================================================================
__global__ void circle_kernel(arm* _arm, circle* _c , double *path ,int *n_path, int dir,  double  e_rad, double spd , bool pos/*位置を優先して終了する*/)
{
    const int nmem = 1024;
    //    printf("in kernel");
//    __shared__ short    result[nmem];                  //結果符号拡張してくれるかしら
    __shared__ short    succeed[nmem];     //成功失敗
    __shared__ bool     fin;                //終了フラグ

    //初速と目標速度との関係で減速するのか、加速するのかを決めます。
    __shared__ bool     acc;                //加速するかどうか
    __shared__ double   spd_span;           //速度の幅
    __shared__ int      default_idx;        //

    //ブロックとスレッドの組み合わせで
    int idx = threadIdx.x;      //512
    int blc = blockIdx.x;       //512

    //初期化
    if (idx == 0) {
        fin = false;        //
        acc = (spd > _arm->wd.spd) ? true : false;      //加減速スイッチです。
        spd_span = fabs(spd - _arm->wd.spd);            //加減速による速度の幅です。（初速度～目標速度間の幅）
        default_idx =   acc ? -1 : (blockDim.x);        //
        *n_path = 0;                                    //データ個数の初期化です。
    }
    __syncthreads();        //あるブロック内部のスレッドの同期です。
    for (int loop = 0 ; fin!=true ;++loop ) {
        if (loop > 500) {
            fin = false;
        }
        //現在のアーム地点から指示速度
        //減速するのはどうするんだろうか
        double dt =dir *  (spd_span / blockDim.x)*(idx+ (acc?1:0) ) / 1000; //1ms分なので、 spd rad/sec -> rad/msec に修正        idx=511で bloclDim.xとなるように
        //加速の場合、idx==511で最高速(spd)にならないといけないので+1.
        //減速の場合、idx==0 で再低速（spdにならないといけないので+0）

        link_stat s = _arm->move_able(_cood(_c->progress_rad(*_arm, dt, dir), _arm->rad + dt  ));
        //    mem[idx] =    _arm->move_able(_cood(_c->progress_rad(*_arm, dt , CIRCLE_DIR_CW) , _arm->rad +dt )).stat==MOV_OK ? 1:0;
//        result[idx] = s.stat;
        succeed[idx] = s.stat == MOV_OK ? idx : default_idx;     //成功したら自分のインデックスを入れる。（リダクションのため）
                                                                                    //※失敗の場合のデフォルト値は、減速の場合には小さいほうが採用されるように大きな値にしておく。
                                                                                    // （idx : 0 - blockDim.x-1 なので、 ）
        //     __device__ __host__ vec2 progress_rad(const vec2 & start, double rad, int dir = 0) const;
//        printf("[%d]:%d %d\n", idx, result[idx], succeed[idx]);
        __syncthreads();        //あるブロック内部のスレッドの同期です。

        //===================================================================================
        //      ここで、一番多く動かせたのを探します。
        //      一番大きく動作できたものを選択して、_armにセットする。
        //===================================================================================
        {   //リダクション。減速と加速で条件が違います。
            //条件分岐が少ないように重複して書いてみる。本当にパフォーマンス変わるのか
            if (acc) {
                for (int i = blockDim.x / 2; i > 0; i /= 2) {
                    if (idx < i && succeed[idx + i] > succeed[idx]) {
                        succeed[idx] = succeed[idx + i];    //半分より大きな部分と比較して大きなほうを
                    }
                    __syncthreads();
                }
            }
            else {
                for (int i = blockDim.x / 2; i > 0; i /= 2) {
                    if (idx < i && succeed[idx + i] < succeed[idx]) {       //減速の場合は小さいほうを選択する。
                        succeed[idx] = succeed[idx + i];    //半分より大きな部分と比較して大きなほうを
                    }
                    __syncthreads();
                }
            }
        }
        //ここで一つも進める候補がみつからない場合には終わります。
        if (succeed[0] == default_idx) {
            printf("no succeed idx(fin)\n");
            goto fin;   //
        }

        //最後、succeed[0]に一番大きな添え字が残っているので、そこへ移動させる。
        if (idx == succeed[0] ) {
//            double dt = (spd / blockDim.x) * idx / 1000; //1ms分なので、 spd rad/sec -> rad/msec に修正
            _arm->move(_cood(_c->progress_rad(*_arm, dt, dir), _arm->rad + dt ));
            printf("%d,t:%lf,spd:%lf,acc%lf,dlt:%lf\n" , loop ,_arm->rad ,  _arm->wd.spd , _arm->wd.acc , fabs(_arm->wd.spd)- spd);
           // 記録します。(dt)
            path[(*n_path)++] = dt;

            //===============================================
            //  終了条件の判定をします。
            //===============================================
            if ( pos ) {
                //位置の場合には、ピッタリとまらないことがあるので、行きすぎたら終わる。
//              //もしくは、次の加速で
                if ((dir == CIRCLE_DIR_CW && _arm->rad > e_rad)
                    || (dir == CIRCLE_DIR_CCW && _arm->rad < e_rad)) {
                    printf("pos toutatu!!!\n");
                    fin = true;
                }
            }
            else {
                //動かしたところで、目標到達速度に達すると終了。目標速度
                if (_equal<float>(fabs(_arm->wd.spd), spd)) {
                    printf("spd toutatu!!!\n");
                    fin = true;
                    //                break;
                }
            }
        }
        __syncthreads();

        //とりあえずここで目標速度に到達したら終了としてみる。
    }   //次にすすみます。
fin:;
    if(idx==0)    printf("[%d]kernel fin\n" ,blockIdx.x);   //一回だけ
}

//カーネル呼び出し
void circle_path()
{
    circle* p_c;
    arm* p_arm;
    //まず一回の計算分を格納するバッファを作ります。
//    char 


    //
    double* p_path;     //GPUに渡す軌跡データ
    int    *p_n_path;     //GPUで計算した軌跡の数
    hipError_t s;

    //==========================================================================
    //  軌跡となる円を設定します。
    //==========================================================================
    circle c(100, 50, 50);


    //==========================================================================
    init_path();                    //軌跡の記録データのセットアップです。
    //==========================================================================


    //試しに、0から90度まで、初速0 - 最高速 90/sec- 終了速度 0 で動ける軌跡を探索する。
    //======= 条件です ==============================
    double s_rad = 0;               //姿勢も角度と同じとしてみます。
    double e_rad = PI/2;            //b
    double spd = PI/2;              //PI/2の速度まで
    //======= 条件です ==============================
    //armサークルオブジェクトをgpuへ転送
    {
        s = hipMalloc((void**)&p_c         ,   sizeof(circle));                                _Assert(s == hipSuccess, "hipMalloc failed!");
        s = hipMemcpy(p_c, (void*)&c, sizeof(circle), hipMemcpyHostToDevice);                 _Assert(s == hipSuccess, "hipMalloc failed!");        //
        s = hipMalloc((void**)&p_arm       ,   sizeof(arm));                                   _Assert(s == hipSuccess, "hipMalloc failed!");        //

        s = hipMalloc((void**)&p_path      ,   sizeof(double) * GPU_PATH_MAX);                 _Assert(s == hipSuccess, "hipMalloc failed!");        //軌跡データ
        s = hipMalloc((void**)&p_n_path    ,   sizeof(int));                                   _Assert(s == hipSuccess, "hipMalloc failed!");        //軌跡データの数
    }
    //経路の加速減速を区間を決めていくか？無理ですよね。

    //どうやるのか。とりあえず、始点から始めて、一区間分で動ける位置を探すのを１スレッドでやるか
//    circle_kernel << <1, 512 >> > (p_arm, p_c, CIRCLE_DIR_CW, e_rad, spd);
    //==========================================================================================================
    //   到達位置から逆に加速させ、目標速度に到達するようにする。
    //==========================================================================================================
    double dec_start_pos;       //減速開始位置です。
    {
        _arm.set(_cood(c.rf(e_rad), e_rad));           //アームを終了位置へセットします。
        s = hipMemcpy(p_arm, (void*)&_arm, sizeof(arm), hipMemcpyHostToDevice);   _Assert(s == hipSuccess, "hipMalloc failed!");
        circle_kernel << <1, 512 >> > (p_arm , p_c , p_path , p_n_path ,  CIRCLE_DIR_CCW, e_rad, spd, false);   //減速開始位置の計算です。
        s = hipGetLastError();        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
        //結果をCPUへ転送します。
        s = hipMemcpy((void*)&_arm, p_arm, sizeof(arm), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMalloc failed!");
        dec_start_pos = _arm.rad;
    }
//    addKernel << <1, size >> > (dev_c, dev_a, dev_b);
    //=====================================================================================
    //  ためしに、減速開始位置から速度を反転させて減速させてみる。
    //=====================================================================================
    //ここで、停止位置から逆方向に加速して、速度が到達したところで、終了とする。
    //ここでの_armの位置が減速開始位置になる。
    //ここでアーム状態を書きもどす。
    //減速させてみる。
//    _arm.stop();            //逆方向に
    if(0)
    {
        _arm.reverse();     //減速開始位置でマイナス方向に加速したので反転します。(目標速度で)

        s = hipMemcpy(p_arm, (void*)&_arm, sizeof(arm), hipMemcpyHostToDevice);   _Assert(s == hipSuccess, "hipMalloc failed!");
        circle_kernel << <1, 512 >> > (p_arm, p_c ,  p_path, p_n_path , CIRCLE_DIR_CW, e_rad, 0,false);        //目標速度を0としてやってみる。
        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
        //データを取り出します。
    }
    //=====================================================================================
    // 初期位置から、減速開始位置まで
    //=====================================================================================
#if 1
    {
        _arm.set(_cood(c.rf(s_rad), s_rad) );           //アームをスタート位置へ移動。
        s = hipMemcpy(p_arm, (void*)&_arm, sizeof(arm), hipMemcpyHostToDevice);   _Assert(s == hipSuccess, "hipMalloc failed!");

        circle_kernel << <1, 512 >> > (p_arm, p_c , p_path, p_n_path , CIRCLE_DIR_CW, dec_start_pos , spd , true);        //目標速度を0としてやってみる。
        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
        //結果をCPUへ移動します。
//  debug(armの状態が知りたければ)
//        s = hipMemcpy((void*)&_arm, p_arm, sizeof(arm), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMalloc failed!");
        //ここでは同じか超えている場合
        append_path(p_path, p_n_path);  //これでコピーします
    }
    //※本当は一つ前がいいが
    //=====================================================================================
    // ここから減速させます。
    //=====================================================================================
    //ひとまず速度０まで落としてみます。
    {
        circle_kernel << <1, 512 >> > (p_arm, p_c , p_path, p_n_path , CIRCLE_DIR_CW, e_rad , 0 ,false );        //目標速度を0としてやってみる。
        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
        //結果をCPUへ移動します。
        s = hipMemcpy((void*)&_arm, p_arm, sizeof(arm), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMalloc failed!");
        append_path(p_path, p_n_path);  //これでコピーします
    }
    //ここで、目的位置

    //ピッタリの場所にいくように低い速度でうごかす最低速度で（即停止できる）
    double min_spd = spd / 400;  //toku ※いつでも止まれる速度を計算する必要がある。(それぞれのリンクの加速度が)
    {
        circle_kernel << <1, 512 >> > (p_arm, p_c, p_path, p_n_path, ( e_rad > _arm.rad) ?  CIRCLE_DIR_CW : CIRCLE_DIR_CCW , e_rad ,min_spd , true );        //目標速度を0としてやってみる。
        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
        //結果をCPUへ移動します。
        s = hipMemcpy((void*)&_arm, p_arm, sizeof(arm), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMalloc failed!");
        append_path(p_path, p_n_path);  //これでコピーします
    }
#endif

    hipFree((void*)p_c);
    hipFree((void*)p_arm);

    //
    //ここで計算した軌跡にそって動作させてみます。
    {

        _arm.set(_cood(c.rf(s_rad), s_rad));           //アームをスタート位置へ移動。
        for (int i = 0; i < _path_idx; ++i) {
            _arm.move(_cood(c.progress_rad(_arm, _path_data[i], CIRCLE_DIR_CW), _arm.rad + _path_data[i]));
            //各関節の指示データを保存していきますか
            _arm.save();        //ファイルに保存してみます。
            ::Sleep(1);
        }
    }
//    draw_thread();
#if 0
    arm
      __host__ __device__ void set(_cood pos) { move(pos, false); stop(); }//

    cood:
    _cood(vec2 v, double _rad, spd_acc _wd = spd_acc()) :vec2(v), rad(_rad), wd(_wd) { ; }

    circle::
    __device__ __host__ vec2 rf(double rad)	const {
        return vec2(rvec2(_r, rad)) + _org;
    }
#endif

    //

    printf("success fin");
    return;

Error:;
    return;

}

//サブルーチン
__device__ __host__ _cood _m(arm* _arm, circle* _c,double dt,int dir,double rate)
{
    _cood m;
    {
        double _t;
        //c[1]の軌道にすすんだあと、c[0]のangleを評価しても_armが円周上にない。
        //現在の制御では、アームがどの円周上でも一様に角度が増加するのでそれを利用する。PI/2以上の場合は円➁になるという判断です。
        if ((_t = ((_arm->rad + dt*rate) - PI / 2)) > 0) {  //円①の軌道上での90度を超える場合、
            //円周２の
            m = _cood(_c[1].rf(PI * 3 / 2 - _t), _arm->rad + (dt * rate));       //アーム角度はそのまま継続して増加させてみます。
            //最終的に
//^^^^^^^^^  角度を足すのをやめてみる。
        }
        else {
            m = _cood(_c[0].progress_rad(*_arm, dt, dir), _arm->rad + dt);
        }
    }
    return m;
}

__global__ void circle_circle_calc_kernel(arm* _arm, circle* _c, int* succeed, int* succeed_blc, int dir, double init_spd, double spd,double rate)
{
    //初速と目標速度との関係で減速するのか、加速するのかを決めます。
    __shared__ bool             acc;                    //加速するかどうか
    __shared__ double           spd_span;               //速度の幅
    __shared__ int              default_idx;            //
    __shared__ int     reduction[N_THREAD];     //

    //ブロックとスレッドの組み合わせで成功失敗の
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    //__shared__メモリの初期化です.各ブロックに
    if (threadIdx.x == 0) {
        acc = (spd > init_spd) ? true : false;    //加減速スイッチです。
        spd_span = fabs(spd - init_spd);          //加減速による速度の幅です。（初速度～目標速度間の幅）
        default_idx = acc ? -1 : (blockDim.x * gridDim.x);        //（成功しない場合に

    }
    __syncthreads();        //あるブロック内部のスレッドの同期です。

#if 1   //debug code(break point用)
    if (blockIdx.x == 468 ) {          //debug stopeer
        reduction[threadIdx.x] = succeed[idx];               //高速にリダクションするためにsharedに入れます。
    }
#endif

    //今回チャレンジする移動距離をdtとします。
    double dt = dir * (spd_span / (N_BLOCK * N_THREAD)) * (idx + (acc ? 1 : 0)) / 1000; //1ms分なので、 spd rad/sec -> rad/msec に修正        idx=511で bloclDim.xとなるように

    //もしここで境界を超える場合には、
    //今回のdtで、円１の範囲を超える場合には、ポインタ
    //_dtは角度、_armのvec2が現在の円周上の座標。
    //進む位置と角度を決定します。
    _cood m;
#if 0
    {
        double _t;
        //c[1]の軌道にすすんだあと、c[0]のangleを評価しても_armが円周上にない。
        //現在の制御では、アームがどの円周上でも一様に角度が増加するのでそれを利用する。PI/2以上の場合は円➁になるという判断です。
        if (  ( _t  = ( (_arm->rad +  dt) - PI/2) ) > 0 )  {  //円①の軌道上での90度を超える場合、
            //円周２の
            m = _cood(_c[1].rf(PI * 3 / 2 - _t), _arm->rad + (dt*1.2) );       //アーム角度はそのまま継続して増加させてみます。
                                                                        //最終的に
                                                //^^^^^^^^^  角度を足すのをやめてみる。
        }
        else {
            m = _cood(_c[0].progress_rad(*_arm, dt, dir), _arm->rad + dt);
        }
    }
#else
    m = _m(_arm, _c, dt, dir,rate);  //cirlce_moveカーネルと処理を共通化するため
#endif

    link_stat s = _arm->move_able(m);


    //※ここは減速でだめになりますので注意
#if 1
 /*   enum {
        //
        MOV_SPD_OVER = 3,		//最大速度を超えて動かそうとした
        MOV_ACC_OVER = 2,		//加速が大きすぎてNG
        MOV_DEC_OVER = 1,		//減速が大きすぎてNG
        MOV_OK = 0,				//成功
    };*/
    //succeedの値には全リンクのステータスを書きます。
    if (s.stat == MOV_OK) {
//        succeed[idx] = s.stat == MOV_OK ? idx : (default_idx * ((s.stat * 10) + s.no));     //成功したら自分のインデックスを入れる
        succeed[idx] = s.stat == MOV_OK ? idx :default_idx ;     //成功したら自分のインデックスを入れる
    }
    else {      //
        //全リンクのNGの理由を調べます。
        link_stat s[3];
        _arm->move_able(m, 1.0,s);
        succeed[idx] = default_idx * ((s[2].stat * 100000 + (s[2].no * 10000)) + (s[1].stat * 1000 + s[1].no * 100) + ((s[0].stat * 10) + s[0].no));
    }
#else
    succeed[idx] = s.stat == MOV_OK ? idx : default_idx ;     //成功したら自分のインデックスを入れる
#endif
    reduction[threadIdx.x] = succeed[idx];               //高速にリダクションするためにsharedに入れます。

    //※失敗の場合のデフォルト値は、減速の場合には小さいほうが採用されるように大きな値にしておく。
    // （idx : 0 - blockDim.x-1 なので、 ）
//     __device__ __host__ vec2 progress_rad(const vec2 & start, double rad, int dir = 0) const;
//        printf("[%d]:%d %d\n", idx, result[idx], succeed[idx]);
    __syncthreads();        //あるブロック内部のスレッドの同期です。
    //==================================================================================
    // リダクションでブロック単位の評価を行います。
    //===================================================================================
    {   //リダクション。減速と加速で条件が違います。
        //条件分岐が少ないように重複して書いてみる。本当にパフォーマンス変わるのか
        int _idx = threadIdx.x;     //ここで使われるのはスレッドのインデックスです。
        if (acc) {
            for (int i = blockDim.x / 2; i > 0; i /= 2) {
                if (_idx < i && reduction[_idx + i] > reduction[_idx]) {
                    reduction[_idx] = reduction[_idx + i];    //半分より大きな部分と比較して大きなほうを
                }
                __syncthreads();
            }
        }
        else {
            for (int i = blockDim.x / 2; i > 0; i /= 2) {
                if (_idx < i && reduction[_idx + i] < reduction[_idx]) {       //減速の場合は小さいほうを選択する。
                    reduction[_idx] = reduction[_idx + i];    //半分より大きな部分と比較して大きなほうを
                }
                __syncthreads();
            }
        }
        //ここでreduction[0]が、このブロック内で一番いい値です。
        if (threadIdx.x == 0) {
            succeed_blc[blockIdx.x] = reduction[0];
        }
    }

}
//成功配列から、一番効率のいいものを選択して実際にアームを動作させる
__global__ void circle_circle_move_kernel(arm* _arm, circle* _c, int* succeed, int* succeed_blc, double* path, int* n_path, int* fin, int dir, double e_rad, double init_spd, double spd, bool pos, double rate,int kernelno = 0)
{
    __shared__ bool             acc;                    //加速するかどうか
    __shared__ double           spd_span;               //速度の幅
    __shared__ int              default_idx;            //
    //ブロックとスレッドの組み合わせで成功失敗の
#if 1       //debug
    if (kernelno == 20) {       //20番カーネルが、arm状態があっているかどうか
        __syncthreads();
    }
#endif

    int idx = threadIdx.x;
    //__shared__メモリの初期化です.各ブロックに
    if (threadIdx.x == 0) {
        acc = (spd > init_spd) ? true : false;                  //加減速スイッチです。
        spd_span = fabs(spd - init_spd);                        //加減速による速度の幅です。（初速度～目標速度間の幅）
        //        default_idx = acc ? -1 : (blockDim.x * gridDim.x);          //（成功しない場合に
        default_idx = acc ? -1 : 0x7fffffff;    //int最大
    }
    __syncthreads();
    //==================================================================================
    // リダクションでブロック単位の評価を行います。
    //===================================================================================
    int adopted_idx;        //これが採用する
    {   //リダクション。減速と加速で条件が違います。
        //条件分岐が少ないように重複して書いてみる。本当にパフォーマンス変わるのか
        int _idx = threadIdx.x;     //ここで使われるのはスレッドのインデックスです。
        if (acc) {
            for (int i = blockDim.x / 2; i > 0; i /= 2) {
                if (_idx < i && succeed_blc[_idx + i] > succeed_blc[_idx]) {
                    succeed_blc[_idx] = succeed_blc[_idx + i];    //半分より大きな部分と比較して大きなほうを
                }
                __syncthreads();
            }
        }
        else {
            for (int i = blockDim.x / 2; i > 0; i /= 2) {
                if (_idx < i && succeed_blc[_idx + i] < succeed_blc[_idx]) {       //減速の場合は小さいほうを選択する。
                    succeed_blc[_idx] = succeed_blc[_idx + i];    //半分より大きな部分と比較して大きなほうを
                }
                __syncthreads();
            }
        }
        //ここでscceed_[0]が、このブロック内で一番いい値です。
        if (threadIdx.x == 0) {
            adopted_idx = succeed_blc[0];   //succeed_blcには、ブロック、スレッドを通したインデックスが入っています。
        }
    }

    //選択されたインデックスでアームを動かし、その結果条件にあっているかを判定します。
    if (threadIdx.x == 0) {
        if (adopted_idx < 0) {  //一つも動かせない場合
            printf("cant move\n");     *fin = -1;       goto error;
        }
        //        if (pos && (*n_path > 105)) {            printf("-");        }
        double dt = dir * (spd_span / (N_BLOCK * N_THREAD)) * (adopted_idx + (acc ? 1 : 0)) / 1000;     //1ms分なので、 spd rad/sec -> rad/msec に修正        idx=511で bloclDim.xとなるように

        _cood m = _m(_arm, _c, dt, dir,rate);  //cirlce_moveカーネルと処理を共通化するため
        _arm->move(m);                            //動かす。
        //      _arm->move(_cood(_c->progress_rad(*_arm, dt, dir), _arm->rad + dt));                            //動かす。
        //        printf("[%d]t:%lf,spd:%lf,acc%lf,dlt:%lf\n", *n_path, _arm->rad, _arm->wd.spd, _arm->wd.acc, fabs(_arm->wd.spd) - spd);
                // 記録します。(dt)
        path[(*n_path)++] = dt;

        //===============================================
        //  終了条件の判定をします。
        //===============================================
        if (pos) {
            //位置の場合には、ピッタリとまらないことがあるので、行きすぎたら終わる。
//              //もしくは、次の加速で
            if ((dir == CIRCLE_DIR_CW && _arm->rad > e_rad)
                || (dir == CIRCLE_DIR_CCW && _arm->rad < e_rad)) {
                printf("pos toutatu!!!\n");
                *fin = (int)true;
            }
            else {
                *fin = (int)false;
            }
        }
        else {
            //動かしたところで、目標到達速度に達すると終了。目標速度
            if (_equal<float>(fabs(_arm->wd.spd), spd, 0.0005)) {   //精度はちょっと甘くしてみます。
                printf("spd toutatu!!! (%d)-(%d)(%d)\n", kernelno, blockIdx.x, threadIdx.x);
                *fin = (int)true;
            }
            else {
                *fin = (int)false;
            }
        }
    }
    return;
error:
    //finに結果が表示されます。
    return;
}

static int __a; //
//異なる円に接続する挑戦をしてみる。
void circle_circle_path()
{
    //    const int threads   = 512;    //
    //    const int blocks    =   128;    //

    circle* p_c;

    arm* p_arm;

    //計算分を格納するバッファを作ります。
    static  int    succeed[N_THREAD * N_BLOCK];       //staticとしてみる。
    static  int    succeed_blc[N_BLOCK];       //staticとしてみる。
    int* p_succeed;                                //デバイス側です。
    int* p_succeed_blc;                          //ブロック単位でのリダクション結果を格納するための

    //経路データです。
    double* p_path;     //GPUに渡す軌跡データ
    int* p_n_path;     //GPUで計算した軌跡の数
    hipError_t s;

    //アームが目的の状態になったかどうか（カーネルが判断した結果を格納します。)
    //int fin;
    int* p_fin;

    //==========================================================================
    //  軌跡となる円を設定します。
    //==========================================================================
    circle c[2] = { circle(100, 50, 50)  , circle(100,50,250)};

    //==========================================================================
    init_path();                    //軌跡の記録データのセットアップです。
    //==========================================================================
    //試しに、0から90度まで、初速0 - 最高速 90/sec- 終了速度 0 で動ける軌跡を探索する。
    //======= 条件です ==============================
    double s_rad = 0;               //姿勢も角度と同じとしてみます。
    double e_rad = PI;              //pi相当の位置まで動かします。
                                    //  PI/2以降は、円2に遷移して、PI/2 - PI を、 
                                    //      3PI/2 - PI までマイナスにすすませるか
                                    //      (3PI/2) - ( Θ  - PI/2)　=>  2PI - Θ かな?
    double spd = PI/4;                //PI/2の速度まで
    //======= 条件です ==============================
    //armサークルオブジェクトをgpuへ転送
    {
        //円をコピーです。
        s = hipMalloc((void**)&p_c, sizeof(c));                               _Assert(s == hipSuccess, "hipMalloc failed!");
        s = hipMemcpy(p_c, (void*)&c, sizeof(c), hipMemcpyHostToDevice);     _Assert(s == hipSuccess, "hipMalloc failed!");
        //アームをコピーです。
        s = hipMalloc((void**)&p_arm, sizeof(arm));                                _Assert(s == hipSuccess, "hipMalloc failed!");        //
        //失敗成功(全データを見られるように)    ※
        s = hipMalloc((void**)&p_succeed, sizeof(succeed));                        _Assert(s == hipSuccess, "hipMalloc failed!");
        s = hipMalloc((void**)&p_succeed_blc, sizeof(succeed_blc));                _Assert(s == hipSuccess, "hipMalloc failed!");
        //ブロック単位での成功失敗を格納するためのメモリです。
//        s = hipMalloc()
        //経路の記録。
        s = hipMalloc((void**)&p_path, sizeof(double) * PATH_MAX);                  _Assert(s == hipSuccess, "hipMalloc failed!");        //軌跡データ
        s = hipMalloc((void**)&p_n_path, sizeof(int));                              _Assert(s == hipSuccess, "hipMalloc failed!");        //軌跡データの数
        //カーネルの判定用。
        s = hipMalloc((void**)&p_fin, sizeof(int));                                 _Assert(s == hipSuccess, "hipMalloc failed!");        //軌跡データの数
    }

    //とりあえず最高速のまま、次の円に接続できるのかを確かめてみる。
    //==========================================================================================================
    //  初期位置から減速位置まで動作させていく。
    //  探索のために、ちょっと行きすぎるようにしてみます。
    //==========================================================================================================
    init_path();        //軌跡はクリアです。
    //
    {   //GPU側の軌跡カウンタクリアと、アームの初期位置のセットです。
        int n_path = 0;
        s = hipMemcpy(p_n_path, (void*)&n_path, sizeof(int), hipMemcpyHostToDevice);     _Assert(s == hipSuccess, "hipMalloc failed!");        //

        //円１からスタートです。
        _arm.set(_cood(c[0].rf(s_rad), s_rad));           //アームを開始位置へセットします。
        s = hipMemcpy(p_arm, (void*)&_arm, sizeof(arm), hipMemcpyHostToDevice);   _Assert(s == hipSuccess, "hipMalloc failed!");
    }
    double rate = 1.1;  //
    printf("start search path...");
    {   //探索
        int i = 0;
        //一つも動けなかった場合にいろいろと振ってみるためのパラメータ
        int retry = 0; 
        //        double ov = fabs( e_pos - dec_start_pos)     ;        //少し行きすぎる場所まで記録する。
        for (int fin = 0; !fin; ++i) {

            if (i >= 1111) {
                fin += 1;
                fin -= 1;
            }

            //一回分のカーネルです。
            {
                circle_circle_calc_kernel << <N_BLOCK, N_THREAD >> > (p_arm, p_c, p_succeed, p_succeed_blc, CIRCLE_DIR_CW, 0, spd,rate);
                hipDeviceSynchronize();    s = hipGetLastError();        if (s != hipSuccess) { printf("kernel failed: %s\n", hipGetErrorString(s));        goto Error; }
            }
            ///break
             if (i>=900) {
                          { s = hipMemcpy((void*)&succeed[0], (void*)p_succeed, sizeof(succeed), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
                          { s = hipMemcpy((void*)&succeed_blc[0], (void*)p_succeed_blc, sizeof(succeed_blc), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
                          s = (hipError_t)1;
             }
                        //ここで各ブロックの集計結果をまとめて、一番いいものを選択し、実際にアームを動作させ、軌跡を記録します。
            {
                //※ちょっと行きすぎる場所まで
                circle_circle_move_kernel << <1, N_BLOCK >> > (p_arm, p_c, p_succeed, p_succeed_blc, p_path, p_n_path, p_fin, CIRCLE_DIR_CW  , e_rad  , 0, spd,true,rate);        //位置を目標に
                hipDeviceSynchronize();        //終了待ち
            }
            //結果をCPUに転送して評価していきます。ここが一番時間がかかる。
            { s = hipMemcpy((void*)&fin, (void*)p_fin, sizeof(int), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }

            if (retry && fin == 1) { //リトライでうまくいった
                retry = 0;
            }
            if (fin < 0) {  //finが－１の場合には進めなかった。
                if (! retry ) {
                    rate = 0.5;         //0.5からスターとです。
                }
                else {
                    rate += 0.01;           //ちょっとづつ角度を振ってみる。
                }
                ++retry;
                --i;        //iはインクリメントしないようにする。
                fin = 0;    //継続するようにします。
            }


            if (i >= 900) {
                { s = hipMemcpy((void*)&_arm, (void*)p_arm, sizeof(arm ), hipMemcpyDeviceToHost);   _Assert(s == hipSuccess, "hipMemcpy failed!");  }
                //このあたりから円➁にさしかかります。
         //                if (!retry) rate = 0.1;     //はじめだけ
            }

        }
        printf("fin(n_path : %d)\n", i-1);
    }
    //軌跡をアップロードします。
    append_path(p_path, p_n_path);  //これでコピーします
    //
    {
        printf("free gpu memory..");
        hipFree((void*)p_c);
        hipFree((void*)p_arm);
        hipFree((void*)p_succeed);
        hipFree((void*)p_succeed_blc);
        hipFree((void*)p_path);
        hipFree((void*)p_n_path);
        hipFree((void*)p_fin);
        printf("fin\n");
    }

    //ここで計算した軌跡にそって動作させてみます。
    {
        printf("arm move..");
        _arm.set(_cood(c[0].rf(s_rad), s_rad));           //アームをスタート位置へ移動。
        for (int i = 0; i < _path_idx; ++i) {
            //            if (i > 962) {                ::Sleep(1);            }
            //これは動かし方が違う。c[1]に接続しないと
//            _arm.move(_cood(c[0].progress_rad(_arm, _path_data[i], CIRCLE_DIR_CW), _arm.rad + _path_data[i]));
            _cood m = _m(&_arm , c, _path_data[i] , CIRCLE_DIR_CW , rate);  //cirlce_moveカーネルと処理を共通化するため
            _arm.move(m);                            //動かす。
            //各関節の指示データを保存していきますか
            _arm.save();        //ファイルに保存してみます。
            //            ::Sleep(1);
        }
        printf("fin\n");
    }

    printf("all fin\n");
    return;
Error:;
    printf("failed");

}


//リダクション.最大のものだけを選択する
__device__
void _do_reduction(int* reduction_arr, int arr_size)
{
    int _idx = threadIdx.x;     //ここで使われるのはスレッドのインデックスです。
    int abre_val = -1;                //あぶれた値。（-1の場合には無効となる。）
#if 1
    for (int i = arr_size / 2; i > 0; i /= 2) {
        if (_idx < i && reduction_arr[_idx + i] > reduction_arr[_idx]) {
            reduction_arr[_idx] = reduction_arr[_idx + i];    //半分より大きな部分と比較して大きなほうを
        }
        //
        if (_idx == 0) {
            if (abre_val != -1) {   //もしあぶれた数がある場合にはそれを再度評価します。
                if (abre_val > reduction_arr[_idx]) {
                    reduction_arr[_idx] = abre_val;
                }
            }
            //次回のあぶれ数を
            if (i & 1) {   //もし奇数の場合には一つあぶれます。
                abre_val = reduction_arr[i - 1];
            }
            else {
                abre_val = -1;
            }
        }
        __syncthreads();
    }
#else
    if (select_max) {
        for (int i = arr_size / 2; i > 0; i /= 2) {
            if (_idx < i && reduction_arr[_idx + i] > reduction_arr[_idx]) {
                reduction_arr[_idx] = reduction_arr[_idx + i];    //半分より大きな部分と比較して大きなほうを
            }
            //
            if (_idx == 0) {
                if (abre_val != -1) {   //もしあぶれた数がある場合にはそれを再度評価します。
                    if (abre_val > reduction_arr[_idx]) {
                        reduction_arr[_idx] = abre_val;
                    }
                }
                //次回のあぶれ数を
                if (i & 1) {   //もし奇数の場合には一つあぶれます。
                    abre_val = reduction_arr[i - 1];
                }
                else {
                    abre_val = -1;
                }
            }
            __syncthreads();
        }
    }
    else {
        for (int i = arr_size / 2; i > 0; i /= 2) {
            if (_idx < i && reduction_arr[_idx + i] < reduction_arr[_idx]) {  //減速の場合は小さいほうを選択する。
                reduction_arr[_idx] = reduction_arr[_idx + i];                //半分より大きな部分と比較して大きなほうを
            }
            if (_idx == 0) {
                if (abre_val != -1) {   //もしあぶれた数がある場合にはそれを再度評価します。
                    if (abre_val < reduction_arr[_idx]) {
                        reduction_arr[_idx] = abre_val;
                    }
                }
                //次回のあぶれ数を
                if (i & 1) {   //もし奇数の場合には一つあぶれます。
                    abre_val = reduction_arr[i - 1];
                }
                else {
                    abre_val = -1;
                }
            }
            __syncthreads();
        }
    }
#endif
    //対象となる配列の数が

}