﻿#include "windows.h"
#include "hip/hip_runtime.h"
#include ""
#include "commonmodule.h"
#include "kernels.h"
#include "draw.h"


#include "inv_kinetic.h"
int main()
{
    device_query();

#if 0
{
      _3link_calc_T<float> calc(230, 210, 144);
      mlti<_Vector3d<float>,2>r = calc.calc_inv_kinetic(220, 320, 1.571);

      //検算する
      _cood p = calc.calc_fwd_kinetic(&r[0][0]);hak
      p = calc.calc_fwd_kinetic(&r[1][0]);

#endif

    log_init();

    //描画スレッドです。
    HANDLE hThread = CreateThread(NULL, 0, draw_thread, NULL, 0, NULL);

//    circle_path();
//    circle_path_2();
      //2つの円を接続
//    circle_circle_path();
//    line_path();
    //ベジェ曲線上の軌跡を書いてみます。
    bezier_path();

    if(hThread) {
        WaitForSingleObject(hThread, INFINITE); // スレッドが終了するのを待つ
        CloseHandle(hThread); // スレッドのハンドルを閉じる
    }
    return 0;
}
