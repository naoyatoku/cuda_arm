#include "hip/hip_runtime.h"
﻿#include "kernels.h"
#include "linear.h"
#include "bezier.h"

//リダクション.最大のものだけを選択する
__device__
void _do_reduction(int* reduction_arr, int arr_size)
{
    int _idx = threadIdx.x;     //ここで使われるのはスレッドのインデックスです。
    int abre_val = -1;                //あぶれた値。（-1の場合には無効となる。）
    for (int i = arr_size / 2; i > 0; i /= 2) {
        if (_idx < i && reduction_arr[_idx + i] > reduction_arr[_idx]) {
            reduction_arr[_idx] = reduction_arr[_idx + i];    //半分より大きな部分と比較して大きなほうを
        }
        //
        if (_idx == 0) {
            if (abre_val != -1) {   //もしあぶれた数がある場合にはそれを再度評価します。
                if (abre_val > reduction_arr[_idx]) {
                    reduction_arr[_idx] = abre_val;
                }
            }
            //次回のあぶれ数を
            if (i & 1) {   //もし奇数の場合には一つあぶれます。
                abre_val = reduction_arr[i - 1];
            }
            else {
                abre_val = -1;
            }
        }
        __syncthreads();
    }
}
/*

enum {
    TYPE_LINEAR = 0,
    TYPE_BEZIER,
};

//デバイスコードで型毎にことなる処理を行うためのしくみ
template<class T>
__device__ int type();
template<>
__device__
int type<linear>(){   return TYPE_LINEAR;}
template<>
__device__
int type<bezier>(){   return TYPE_BEZIER;}



__host__
static bool _check_kernl_error()
{
    hipError_t s;
    s = hipGetLastError();
    if (s != hipSuccess) {
        printf("kernel failed: %s\n", hipGetErrorString(s));
        return false;
    } //
    return true;
}

//共通で使えるCUDAコードをおいておきます。
template<class T>
__device__
bool check_condition(const arm* _arm , float tgt_spd , int dir , _cood *tgt_pos , int fin_condition)
{
    switch(fin_condition){
    case FIN_SPD:
        if (_equal<float>(_arm->d.spd, tgt_spd,0.1) == true) {
            return true;
        }
        break;
    case  FIN_POS_PASS:
        switch(type<T>()){
            case TYPE_LINEAR:
                if(dir == linear::DIR_P){   //プラス方向に進行している場合、
                    if( _arm->x >= tgt_pos->x ){
                        *fin=true;
                    }
                }else{
                    if( _arm->x <= tgt_pos->x ){
                        *fin=true;
                    }
                }
                break;
            case TYPE_BEZIER:
                //ベジェの場合はt==1.0になったらおしまいだけど、ぴったりにならないきもする
                if(_arm->read_add_info<bezier_pos>().t >= tgt_pos->read_add_info<bezier_pos>().t){  //目標位置より進んでいたら
                    *fin = true;
                }
                break;
        }
        break;
        default:
            _Assert(0,"check_condition():unknown fin_condition\r\n");break;
    }
    return false;
}




//各軌跡関数に応じたターゲット位置の計算です。
template<class T>
__device__ inline
static _cood _tgt_cood(const arm* _arm, const void* p_path_class , float tgt_spd, int dir, int idx, int n_all_thread)
{
    float d_spd = tgt_spd - _arm->d.spd;                             //目標速度にむけての加速度です。(mm/sec)
    //-----------------------------------------------------------------------------------------------------------------
    //  ※toku 
    //  ここは、現在速度～目標速度の間だけだと、減速が効かないので、0もしくは、あるていどの減速までを範囲に含める必要がある。
    //-----------------------------------------------------------------------------------------------------------------
    //　(現在速度＋目標速度への加速度)/1000 (1msあたりに換算)
    float dx = ((d_spd / n_all_thread) * idx) / 1000; //
    //デバッグしやすいように計算を分けます。
    dx += _arm->d.spd / 1000;   //現在速度で1mあたりに進む距離
    //dxこれが現在速度からの加速分です。現在速度の
    _cood c;
    //仮想関数が使えないのでここは枝分かれです。
    switch (type<T>()) {
    case TYPE_LINEAR:
        c = _cood( ((const linear*)p_path_class)->progress(*_arm,dx,dir) , _arm->rad);   //デバッグ用に変数にします。    
        break;

    case TYPE_BEZIER:
        bezier_pos pos;
        c = _cood ( ((const bezier*)p_path_func)->progress(GPU, _arm->read_add_info<bezier_pos>(), dx, pos), _arm->rad);   //デバッグ用に変数にします。
        //cにベジェの位置情報を付加します。
        c.write_add_info<bezier_pos>(pos);
        break;
    }
    return c;
}


template<class T>
__global__
void _calc_kernel(arm* _arm, const void* _p_path_class , int* block_result, float tgt_spd, int dir)
{
    __shared__ extern int _reduction[N_THREAD];        //int*512 -> 2kbyte

    //ブロックとスレッドの組み合わせで成功失敗の
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;                                          //
    link_stat s = _arm->move_able(_tgt_cood<T>(_arm, _p_path_class , tgt_spd,dir, idx, N_ALL_THREAD));     //

    _reduction[threadIdx.x] = s.stat == MOV_OK ? idx : -1;     //成功したら自分のインデックスを入れる失敗したら
    __syncthreads();        //あるブロック内部のスレッドの同期です。
    //==================================================================================
    // リダクションでブロック単位の評価を行います。
    //===================================================================================
    _do_reduction(_reduction, blockDim.x);
}

template<class T>
__global__
void _move_kernel(arm* _arm , const void* _p_path_class ,int* block_result , float tgt_spd ,_cood *tgt_pos ,  int dir, bool* fin,arm* _path, int* _path_idx , int fin_condition)
{
    //まずブロック毎のデータの中でさらに一番いいものを選択する。
    _do_reduction(block_result , blockDim.x );

    if(threadIdx.x==0){ //
        _gpuAssert(block_result[0] >= 0, "move_kernel():no available result\r\n");       //有効なインデックスがなかった場合はエラーです。
        _cood tgt =_tgt_cood<T>( _arm , _p_path_class , tgt_spd,dir,block_result[0],N_ALL_THREAD);
        _arm->move(tgt);                                   //動かす。
        _path[(*_path_idx)++] = *_arm;    //アーム記録してもいいかも

        if (*_path_idx > MAX_PATH-1) {  //overflow
            printf("\r\npath overflow\r\n");
            *fin = true;
            goto _fin;
        }
        *fin = check_condition<T>(_arm, tgt_spd, dir, tgt_pos, fin_condition);
    }
_fin:
    __syncthreads();    //これいらないか？    
}

template<class T>
__host__
bool _kernel(arm * _arm , void  *_p_path_class , float tgt_spd, int dir , _cood *tgt_pos,arm *_path , int *_path_idx , int fin_condition)
{
    const dim3 threads( N_THREAD);
    const dim3 blocks(N_BLOCK);
    cpu_gpu_mem<int>	block_result(N_BLOCK);		//これが全部の結果を保持するようにします。一度の
    cpu_gpu_mem<bool>fin;

    esc_clr();  //画面はクリアする。


    //※アーム状態はカーネル呼び出し前に設定してください。
    for (fin(CPU) = false;  fin(CPU) != true; fin.Transfer_to_CPU() ) {      //この処理に時間がかかるのでこのループ自体も
        _calc_kernel<T> << <blocks, threads >> > ( _arm , _p_path_class , block_result.gpu , tgt_spd, dir);
        hipDeviceSynchronize();
        if (_check_kernl_error() != true) {
            goto _error;
        }
        //この時点でベストが見つかっているとする。
        _move_kernel<T> << <1, blocks >> > (_arm , _p_path_class , block_result.gpu , tgt_spd, tgt_pos , dir , fin.gpu , _path , _path_idx ,fin_condition );
        hipDeviceSynchronize();
        if (_check_kernl_error() != true) {
            goto _error;
        }
    }

    return true;
_error:
    return false;
}
*/