#include "hip/hip_runtime.h"
﻿#include "kernels.h"
#include "draw.h"
#include "stdio.h"
//リダクション.最大のものだけを選択する
__device__
void _do_reduction(int* reduction_arr, int arr_size)
{
    int _idx = threadIdx.x;     //ここで使われるのはスレッドのインデックスです。
    int abre_val = -1;                //あぶれた値。（-1の場合には無効となる。）
    for (int i = arr_size / 2; i > 0; i /= 2) {
        if (_idx < i && reduction_arr[_idx + i] > reduction_arr[_idx]) {
            reduction_arr[_idx] = reduction_arr[_idx + i];    //半分より大きな部分と比較して大きなほうを
        }
        //
        if (_idx == 0) {
            if (abre_val != -1) {   //もしあぶれた数がある場合にはそれを再度評価します。
                if (abre_val > reduction_arr[_idx]) {
                    reduction_arr[_idx] = abre_val;
                }
            }
            //次回のあぶれ数を
            if (i & 1) {   //もし奇数の場合には一つあぶれます。
                abre_val = reduction_arr[i - 1];
            }
            else {
                abre_val = -1;
            }
        }
        __syncthreads();
    }
}

//
__host__
void _dump_path(cpu_gpu_mem<arm>& path, cpu_gpu_mem<int>& path_idx, bool disp_link/*=false*/)
{
    path.Transfer_to_CPU();
    path_idx.Transfer_to_CPU();
    for (int i = 0; i < path_idx(CPU); ++i) {
        const arm& cur = path(CPU, i);
        printf("[%d](%f,%f,%f)spd(%f)\r\n", i, cur.x, cur.y, cur.rad, cur.d.spd);
        if (disp_link) {
            for (int l = 0; l < 3; ++l) {
                printf("        L%d(pos[%f(deg:%f)]spd[%f]\r\n", l, cur.lnk(l).r, RAD2DEG(cur.lnk(l).r), cur.lnk(l).d.spd);
            }
        }
    }
}
static int __a; //デバッグ用
__host__
void _draw_path(cpu_gpu_mem<arm>& path, cpu_gpu_mem<int>& path_idx, int wait/*=1*/)
{
    path.Transfer_to_CPU();
    path_idx.Transfer_to_CPU();
    for (int i = 0; i < path_idx(CPU); ++i) {
        const arm& cur = path(CPU, i);
        draw(cur);
//        ::Sleep(wait);
        //ウェイトを
        for (int i = 0; i < (100000)*wait; ++i) {
            __a++;
        }
//        printf(".");
    }
}



template<>
__device__
int type<linear>() { return TYPE_LINEAR; }
template<>
__device__
int type<bezier>() { return TYPE_BEZIER; }
