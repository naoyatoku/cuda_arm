#include "hip/hip_runtime.h"
﻿#include "kernels.h"

//リダクション.最大のものだけを選択する
__device__
void _do_reduction(int* reduction_arr, int arr_size)
{
    int _idx = threadIdx.x;     //ここで使われるのはスレッドのインデックスです。
    int abre_val = -1;                //あぶれた値。（-1の場合には無効となる。）
    for (int i = arr_size / 2; i > 0; i /= 2) {
        if (_idx < i && reduction_arr[_idx + i] > reduction_arr[_idx]) {
            reduction_arr[_idx] = reduction_arr[_idx + i];    //半分より大きな部分と比較して大きなほうを
        }
        //
        if (_idx == 0) {
            if (abre_val != -1) {   //もしあぶれた数がある場合にはそれを再度評価します。
                if (abre_val > reduction_arr[_idx]) {
                    reduction_arr[_idx] = abre_val;
                }
            }
            //次回のあぶれ数を
            if (i & 1) {   //もし奇数の場合には一つあぶれます。
                abre_val = reduction_arr[i - 1];
            }
            else {
                abre_val = -1;
            }
        }
        __syncthreads();
    }
}


template<>
__device__
int type<linear>() { return TYPE_LINEAR; }
template<>
__device__
int type<bezier>() { return TYPE_BEZIER; }
