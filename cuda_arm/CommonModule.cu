#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------
//	�ėp�I�Ȋ֐��Q�ł��B
//---------------------------------------------------------------------------
#include	<windows.h>
#include	<stdio.h>
#include	<varargs.h>	
#include "CommonModule.h"

#include "string.h"

//toku �ȉ� ���ʎqLs���݂���܂���̑Ώ�
#include <locale.h>
//#include<stdio.h>
//#include<wchar.h>
//#include<tchar.h>


static bool debug=false;
//------------------------------------------------------------------------------------------------------
//	�N�I�[�e�[�V�����𐔂��郂�W���[��
//	�������]�����āA�w��̃N�I�[�e�[�V���������łƂ܂��Ă��邩�ǂ���
//	�𔻒肵�܂��B
//	�N�I�[�e�[�V�����̏�ԃo�b�t�@���w�肷��΁A�N�I�[�e�[�V�����󋵂�
//	�񍐂��܂��B�O��̏�Ԃ������Ɏw�肷��ƁA���̑����Ƃ��Đ����܂��B
//
//	����
//	const char	*str		�E�E�E		�]�����镶����
//	const char	*quotes		�E�E�E		�N�I�[�e�[�V�����̑g�ݍ��킹�ł��B�񕶎���g�ł��B
//	int			*quotestat	�E�E�E		�N�I�[�e�[�V�����̏�Ԃł��Bint �~ QUOTE_MAX�̃o�b�t�@��p�ӂ��Ă��������B
//										NULL�ł������ł��B
//										����Ƀo�b�t�@���w�肷��΁A�O�񂩂�̑����Ƃ��ăN�I�[�e�[�V�����𐔂��A�ŏI�I��
//										��Ԃ������߂��܂��B
//	int ofs					�E�E�E		���������������Ă����NULL�I�[�łȂ��A�擪�`ofs�܂ł̕������]�����܂�
//	�߂�l
//		BOOL	�����񂪃N�I�[�e�[�V�����̑����łƂ܂��Ă��邩�ǂ���
//---------------------------------------------------------------------------------------------------------------------
#define QUOTE_MAX	12		//�N�I�[�e�[�V�����̍ő�l�ł��B
bool isinQuote(const char *str ,  const char* quote , int ofs=0)
{
	//�N�I�[�e�[�V������Ԃł��B
	int quotestat[QUOTE_MAX];	memset(quotestat , 0 , sizeof(quotestat));

	//�y�A�ɂȂ��Ă��Ȃ��ꍇ�̓G���[�ł��B
	_Assert( (strlen(quote) % 2) == 0	,	"isinQuote::not avail quote : %s\r\n"			,	quote);

	//�N�I�[�e�[�V�����𐔂��܂��B
	//
	for(int p=0 ; *(str+p) != '\0' ; p++ ) {
		int i;	
		for( i = 0 ; quote[i]!='\0' ; i+=2 ) {
			if( quote[i]==quote[i+1] ) {	//�N�H�[�e�[�V�����������ꍇ�̓l�X�g�Ή����Ȃ�(���ڂɏo�Ă�����A����́u���v)
				if(*(str+p) == quote[i])	quotestat[i] = quotestat[i] ? 0 : 1;	//�P���ȃg�O���ł��B
			} else {
				if		(	*(str+p)==quote[i]		)	quotestat[i]++;
				else if	(	*(str+p)==quote[i+1]	)	quotestat[i]--;
				//�����ԈႢ��0�ȉ��ɂȂ��Ă��܂����ꍇ�͂���͐����Ȃ����Ƃɂ��܂��B(�����Ȃ�I�[�������ꍇ�Ȃ�)
				if(quotestat[i] < 0) quotestat[i]=0;
			}
		}
		//����ofs�w�肪����΂����̕����܂łł��B
		if(ofs) {	if( p >= ofs ) break;		}
	}
	//���݃N�I�[�e�[�V�����̒��ɂ�����̂��ǂ����𔻒肵�܂��B
	{
		int i;
		for(i=0 ; quote[i]!='\0' ; i++) if(quotestat[i]) return true;
	}
	return false;
}

//-------------------------------------------------------------
//	�w�肳�ꂽ�������A�w�肳�ꂽ������̒��Ɋ܂܂��
//	�����ƈ�v���邩�ǂ���
//-------------------------------------------------------------
bool matchStr(char c , const char *str)
{
	int i;
	for(i= 0 ; *(str+i) != '\0' ; i++ ) {
		if( c == *(str+i) )  {
			return true;
		}
	}
	return false;
}



//-------------------------------------------------------------
//	�f���~�^��������(�����p�^�[���w���)
//	����
//		char *			buf				�e�L�X�g�o�b�t�@�iNULL�I�[�K�v�j
//		const char*		find_delm		�f���~�^������
//		const char*		str_pattern		�����ΏۂƂȂ镶����
//
//	�߂�l
//		�����������_�ł̃|�C���^��Ԃ��܂��B
//
//	����
//		str_pattern�Ɋ܂܂�镶���������ԁA�������܂��B
//		���̊ԁAfind_delm������ɓ����Ă��镶�������������_��
//		�����I�����āA���̃|�C���^��Ԃ��܂��B
//
//		����str_pattern�Ɋ܂܂�Ȃ������񂪏o�Ă�����Anull���o�Ă����肷���
//		�����Ō����I���ƂȂ�ANULL��Ԃ��܂��B
//		str_pattern��NULL�̏ꍇ�́A�����p�^�[���̕]���͂����ANULL�I�[�݂̂̕]���ɂȂ�܂��B
//-------------------------------------------------------------
char *_findDelm(const char *buf , const char *delm ,const char *str_pattern )
{
	int i=0;
	//�܂��f���~�^�������܂ŃT�[�`���܂��B
	for( ; ; i++ )	{
		if( matchStr( *(buf+i) , delm ) )	break;			//delm����v���܂����B
		if(	*(buf+i) == '\0' )				return (char*)NULL;	//�����炸�ɁA�����ɂȂ�܂����B
		//�w�蕶���p�^�[��������΁A���̕����łȂ��Ȃ������_�ŏI���ł��B
		if( str_pattern ) {
			if( ! matchStr( *(buf+i) , str_pattern) ) return (char*)NULL;
		}
	}
	return (char*)(buf+i);
}

//-------------------------------------------------------------
//	�f���~�^��������
//	����
//		const char *buf		:	�e�L�X�g�o�b�t�@�iNULL�I�[�K�v�j
//		const char *delm	:	�f���~�^������
//	�߂�l
//		�w��f���~�^(������̂����ǂꂩ�ꕶ��)�������������_�̃|�C���^��Ԃ��܂��B
//		�����f���~�^�������炩�����ꍇ��NULL�|�C���^��Ԃ��܂��B
//-------------------------------------------------------------
char *findDelm(const char *buf , const char *delm)
{
	return ( _findDelm( buf , delm , (const char*)NULL ));
}


//-------------------------------------------------------------
//	�t����Delm��������p�^�[���ł�
//-------------------------------------------------------------
char *findDelm_rev(const char *buf , const char *delm)
{
	int i;
	for( i = strlen(buf) - 1 ; i >= 0 ; i--) {
		if(	matchStr( buf[i] , delm) ) return (char*)&(buf[i]);
	}
	return (char*)NULL;
}

//-------------------------------------------------------------
//	�f���~�^���Ƃ��肷����
//	����
//		const char *buf		:	�e�L�X�g�o�b�t�@�iNULL�I�[�K�v�j
//		const char *delm	:	�f���~�^������
//	�߂�l
//		�w��f���~�^�łȂ���������������|�C���^��Ԃ��܂�
//-------------------------------------------------------------
char *overDelm(const char *buf , const char*delm)
{
	int i=0;
	//�܂��f���~�^�������܂ŃT�[�`���܂��B
	for( ; ; i++ )	{
		if( !matchStr( *(buf+i) , delm ) )	break;			//delm�łȂ������������܂���
		if(	*(buf+i) == '\0' )				return (char*)NULL;	//�����炸�ɁA�����ɂȂ�܂����B
	}
	return (char*)(buf+i);
}
//�f���~�^���Ƃ��肷����(������̍Ōォ��t�ɒT���o�[�W�����ł�)
char *overDelm_rev(const char *buf , const char*delm)
{
	int i;
	for( i = strlen(buf) - 1 ; i >= 0 ; i -- ){
		if( !matchStr( *(buf+i) , delm ) )	break;			//delm�łȂ������������܂���
	}
	if(i<0) return (char*)NULL;	//������Ȃ�����

	return (char*)(buf+i);
}

//-------------------------------------------------------------
//	�f���~�^�������Ă���ɒʂ�߂����ꏊ�̃|�C���^��Ԃ�
//-------------------------------------------------------------
char *findoverDelm(const char *buf , const char *delm)
{
	char *p;
	if( (p = findDelm( buf , delm))!=NULL ) return( overDelm(p , delm) );
	return (char*)NULL;
}

//-------------------------------------------------------------
//	�A�X�L�[�����ȊO�̕�������΂��܂��B
//-------------------------------------------------------------
char *overNoAscii( char *buf )
{
	for( ; *buf != 0  ; buf++ ) {
		if( *(unsigned char*)buf <= 0x7f ) return buf;
	}
	return 0;
}

//-------------------------------------------------------------
//	�o�b�t�@����A�f���~�^�܂œǂݍ���
//	�Ƃɂ����f���~�^�܂œǂ݂܂��B
//
//	�߂�l�F�f���~�^�i�܂��͕�����I�[�j���o�Ă����Ƃ���̃|�C���^��Ԃ��܂��B
//			���������ꕶ�����ǂ߂Ȃ������ꍇ�́A�k���|�C���^��Ԃ��܂��B
//-------------------------------------------------------------
char *getElem(const char*src , char*dst ,const char *delm , int dstmax )
{
	//�܂��f���~�^�͔�΂��܂��B
	if( ! (src = overDelm( src , delm)) ) goto Error;

	//�f���~�^���o�Ă���܂ŕ����R�s�[
	int si, di;
	si = di = 0;
	for(	;	; si++ , di++ ) {
		if(dstmax)	_Assert( di < dstmax, "getElem : buf overflow src=%s , max=%d" , src , dstmax);	//�o�b�t�@�I�[�o�[�t���[�`�F�b�N

		if(  *(src+si) == '\0' ){ if(!si) { goto Error;}  else{	break;}	}//�����������Ȃ��Ă��܂��܂����B
		if( matchStr ( *(src+si) , delm ) ) break;
		*(dst+di) = *(src+si);
	}

	*(dst + di) = '\0';	//�Ō�ɏI�[�����܂��B

	return (char*)(src + si);
Error:
	return (char*)0;
}
//---------------------------------------------------------------------------------------
//	�N�H�[�e�[�V�����̒��̃f���~�^�`�F�b�N�����Ȃ�getElem
//	�N�H�[�e�[�V�����́A������ł��B
//	"\"\"()"�ȂǓ񕶎��łЂƂ̃y�A�Ƃ��Ă��������B�i�K���y�A�Ŏw�肵�Ă��������B���Ȃ���Η�O�Ŏ~�߂܂��B�j
//---------------------------------------------------------------------------------------
char *getElem_withoutQuotation(const char*src , char*dst ,const char *delm , char *quote , int dstmax )
{
	//�����͓����|�C���^�ł�
	char *pdst;
	const char*psrc;
	for( pdst=dst , psrc=src ; ; ) {
		//�l�������܂��B
		{
			char *p;
			if( (p = getElem( psrc , pdst , delm ,dstmax)) == 0 ) {	//������Ȃ������B
				if(psrc!=src)	break;				//��x�ł�getElem�ł��Ă���ꍇ�́A�O��̒l��Ԃ��܂��B
				else			return 0;			//�����Ȃ茩����Ȃ��ꍇ��NULL�ł��B
			}
			psrc = p;	//������΁A�ʒu���X�V���܂��B
		}
//_printf("[getalem_quo: getelem[%s]\r\n" , dst);
		//����src�̓f���~�^�̂Ƃ���łƂ܂��Ă��āA�Ȃ�����dst�ɂ���̓R�s�[����Ă܂���B�i�P��������Ȃ�)
		if( isinQuote( dst ,  quote ) ){		//�擾����������́A�N�I�[�e�[�V�����̓r���Ŏ~�܂��Ă��܂��B������ǂނƂɂ��܂�
			pdst		+=	strlen(pdst);		//�������݃|�C���^��i�߂܂��B
			*pdst++		=	*psrc++;			//�������f���~�^�܂ŃR�s�[���܂��B
			*pdst = '\0';						//NULL�I�[�����̏����ŏ����Ă��܂��̂ŕt�������܂��B
			continue;
		}
		break;
	}

	return (char*)psrc;
}

//-------------------------------------------------------------
//	�R�����g�A�E�g����
//-------------------------------------------------------------
char *CommentOut(char *buf , const char *s , const char *e)
{
	for( ; ; ) {
		//�z���C�g�X�y�[�X�������܂��B
		if(	( buf = overDelm( buf ,	" \t\r\n"))	==	0 ) goto Error;

		//�R�����g�s���ǂ����𔻒f���܂��B�R�����g�s�łȂ��Ȃ�΁A�I���ł��B
		if( strncmp( buf , s  , strlen(s) ) != 0 )		break;

		//�R�����g�X�^�[�g�ł��ΏI�[�������T���܂��B
		for( ; ; buf++ ) {
			if( ! *buf )								goto Error;	//�I�[�ɂȂ��Ă��܂��܂���
			if( strncmp(buf , e , strlen(e)) == 0 )		break;		//�R�����g�̍Ōオ������܂���
		}
		//����buf�̓R�����g�I���̕�����e�̐擪�ɂ��܂��B������΂����Ƃ���̃|�C���^��Ԃ��܂��B
		buf = overDelm(buf , e);
	}
	return buf;

Error:
	return 0;
}

//-------------------------------------------------------------
//	�N�I�[�e�[�V�������͂����ĕ��������蒼���܂�
//-------------------------------------------------------------
void removeQuotation( char *str , char quotation )
{
	if( *str != quotation ) return ;	//�N�H�[�e�[�V�������Ȃ��ꍇ�͕Ԃ��܂�
	for(  ; *(str+1) != quotation	; str++ ) 	*str = *(str+1);
	*str = '\0';
}



//===============
//	log �֘A
//===============
#include "csv.h"
#define		DISP_ROWS	30		//50�s���炢������
#define		BUFFSIZE	256		//�P�s������ł��B

static	char			__str[DISP_ROWS][BUFFSIZE];		//�\�����e��\��
static	int				__i_str;
static	bool			__looped;
static _LOG_CALLBACK	__log_callback;			//���O�������݂��s�����Ƃ��ɃR�[���o�b�N���܂��B

static	csv _log("log.txt",true);	//


void log_init(_LOG_CALLBACK callback)
{
	for (int i = 0; i < DISP_ROWS; ++i) {
		sprintf_s( &__str[i][0] , BUFFSIZE , "[%d]" ,i);
	}
	__i_str = 0;	//
	__looped = false;
	if(callback){
		__log_callback = callback;
	}
}
//�����ό����ɂ��邩
void write_log(const char*fmt,...)
{
//	static char _buf[2*1024];				//����p�̃o�b�t�@�ł��B�񓯊�������Ƃ�΂��ł��B
	va_list ap;	va_start( ap , fmt );		//�����WINDOWS���������B�B�B
	//�Ȃ��������ŗ�O���łĂ��܂��B
	vsprintf_s( &__str[__i_str][0]  ,BUFFSIZE , fmt , ap);
//	sprintf(&__str[__i_str][0], fmt, ap);
	//�����Ńt�@�C���ɂ��������ނ悤�ɂ��܂��B�t�@�C���������݂̂Ƃ��ɂ͊J�Ƃ����܂��B
	{	//�t�@�C���ɂ́A�������t�^���܂��B
		_SYSTEMTIME t;::GetLocalTime(&t);
		_log.writef("[%d.%d.%d %02d:%02d]\t%s\n", t.wYear,t.wMonth,t.wDay,t.wHour,t.wMinute  , &__str[__i_str][0]);
	}
	va_end(ap);
//	strcpy_s(&__str[__i_str][0], BUFFSIZE, str);
	//�\���ʒu���C���N�������g���܂��B
	if (++__i_str > (DISP_ROWS-1)) {
		__looped = true;
		__i_str = 0;
	}
//	log()
//	::SendMessage(__hWnd, WM_USER + 1, 0, 0);
	if(__log_callback){
		__log_callback();
	}
	//�����ɁA���O�t�@�C���ɂ������Ă����܂��B
	//�������݈ʒu�ɏ������ށB
}
//���t�����Ȃ��o�[�W����
void write_log_rare(const char* fmt, ...) {
	va_list ap;	va_start(ap, fmt);		//�����WINDOWS���������B�B�B1222222222222^
	vsprintf_s(&__str[__i_str][0], BUFFSIZE, fmt, ap);
	//�����Ńt�@�C���ɂ��������ނ悤�ɂ��܂��B�t�@�C���������݂̂Ƃ��ɂ͊J�Ƃ����܂��B
/* {	//�t�@�C���ɂ́A�������t�^���܂��B
		_SYSTEMTIME t; ::GetLocalTime(&t);
		_log.writef("[%d.%d.%d %02d:%02d]\t%s\n", t.wYear, t.wMonth, t.wDay, t.wHour, t.wMinute, &__str[__i_str][0]);
	}*/
	_log.write("\s", &__str[__i_str][0] );
	va_end(ap);
	//	strcpy_s(&__str[__i_str][0], BUFFSIZE, str);
		//�\���ʒu���C���N�������g���܂��B
	if (++__i_str > (DISP_ROWS - 1)) {
		__looped = true;
		__i_str = 0;
	}
	//	log()
	//	::SendMessage(__hWnd, WM_USER + 1, 0, 0);
	if (__log_callback) {
		__log_callback();
	}
	//�����ɁA���O�t�@�C���ɂ������Ă����܂��B
	//�������݈ʒu�ɏ������ށB
}

const char*get_log(int idx)
{
	//�Ƃ肠����
	int i = __looped ? __i_str  : 0;	//��x���[�v�������Ƃ́A�\���̃C���f�b�N�X�̍ŏ��͏������݃|�C���^��
	return &__str[  (i + idx)%DISP_ROWS  ][0];
			//
//		if (!__looped && (row == __i_str)) {	//�����o�b�t�@���ꏄ���ĂȂ���΁A�������݈ʒu�ɓ��B�������_�ŏI���ł��B
//			break;
//		}	

}
//==================================================================================================================
//	last_error�Ȃ�
//==================================================================================================================
const char* _error_msg(DWORD msg_id, char *msgcpy,int msgcpy_size)
{
	//	printf("1(id=%d)\n", msg_id);
	LPVOID buffer;DWORD n;
	if((n=FormatMessageA(FORMAT_MESSAGE_ALLOCATE_BUFFER | FORMAT_MESSAGE_FROM_SYSTEM | FORMAT_MESSAGE_IGNORE_INSERTS,
		NULL,
		GetLastError(),
		MAKELANGID(LANG_ENGLISH, SUBLANG_DEFAULT),
		(LPSTR)&buffer, 0, NULL))==0){
//			::OutputDebugStringA("format message fin\n");
		};
	//	wprintf(L"[%s]", (LPCTSTR)buffer);
	//�ԍ����o��
/*	wchar_t buf[64];
	wsprintf(buf, L"lasterr %d [", ::GetLastError());
	::OutputDebugString(buf);	::OutputDebugString((LPWSTR)buffer);	::OutputDebugStringA("]\n");*/
	//
	if(msgcpy){
		strcpy_s(msgcpy,msgcpy_size,(const char*)buffer);
	}
	LocalFree(buffer);
	return msgcpy;
}
/*
const char*LastErrorMsg(void) {
	static char str[256];
	_error_msg(::GetLastError(),str,sizeof(str));
	//���s����������΂���͂̂����܂��B
	for (int i = 0; str[i] != '\0';++i) {
		if ((str[i] == '\r') || (str[i] == '\n')) {
			str[i] = '\0';break;
		}
	}
	return str;
}
*/
//static 
//int Error(const char*str)
int		Error(const char*fmt, ...)
{
	//	printf("\n ==== Eror(%s) last error =====\n" ,str);
	//	LastErrorMsg();
//	tokutoku
	va_list ap;	va_start( ap , fmt );		//�����WINDOWS���������B�B�B
		static char	_buf[256];	//write_log�̃o�b�t�@�ɍ��킹�Ă����܂��B
		_buf[0] = 'E'; _buf[1] = '>';		//�ŏ���E:�Ƃ��Ă݂�B
		vsprintf_s(&_buf[2], sizeof(_buf), fmt, ap);
		write_log(_buf);
	va_end(ap);
	return -1;
}

//assert���܂�
void _Assert_log( bool a, const char*fmt,...){
	if(!a){
		Error(fmt);	
//		draw("ASSERT: [%s]", str);
		//
//		dump(str);
//		for (;;);
		exit(0);
	}
}
void _Assert(bool a, const char*fmt,...)
{
	va_list ap;	va_start(ap, fmt);		//�����WINDOWS���������B�B�B
	if (!a) {
//		Error(false, fmt)
		printf(fmt,ap);
		exit(0);
	}
	va_end(ap);

}



#include <iostream>
#include <hip/hip_runtime.h>
// Converts the version number of the device into the number of cores
// Note: This is a simple version, and newer architectures may not be covered.
int _ConvertSMVer2Cores(int major, int minor) {
    typedef struct {
        int SM;
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] = {
        {0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        {0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
        {0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        {0x37, 192}, // Kepler Generation (SM 3.7) GK21x class
        {0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
        {0x52, 128}, // Maxwell Generation (SM 5.2) GM20x class
        {0x53, 128}, // Maxwell Generation (SM 5.3) GM20x class
        {0x60,  64}, // Pascal Generation (SM 6.0) GP100 class
        {0x61, 128}, // Pascal Generation (SM 6.1) GP10x class
        {0x62, 128}, // Pascal Generation (SM 6.2) GP10x class
        {0x70,  64}, // Volta and Turing Generation (SM 7.0) GV10x class
        {0x72,  64}, // Turing Generation (SM 7.2) TU10x class
        {0x75,  64}, // Turing Generation (SM 7.5) TU10x class
        {0x80, 128}, // Ampere Generation (SM 8.0) GA10x class
        {0x86, 128}, // Ampere Generation (SM 8.6) GA10x class
        {-1, -1}
    };

    int index = 0;
    while (nGpuArchCoresPerSM[index].SM != -1) {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }
    // If we don't find the values, we default to return -1
    return -1;
}

int device_query()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount); // ���p�\�ȃf�o�C�X�̐����擾

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device); // device�ԍ��̃f�o�C�X�̃v���p�e�B���擾

        printf("\nDevice %d: \"%s\"\n", device, deviceProp.name);

        // Compute capabilities
        printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

        // Amount of global memory
        printf("  Total amount of global memory:                 %.2f MBytes\n", (float)deviceProp.totalGlobalMem / 1048576.0f);

        // Number of multiprocessors
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
        printf("  max threads per multiprocessors:               %d\n", deviceProp.maxThreadsPerMultiProcessor);
//        printf("  ==>cuda core total                             %d\n", deviceProp.multiProcessorCount * deviceProp..maxThreadsPerMultiProcessor);

        // Maximum number of threads per block
        printf("  Max number of threads per block:               %d\n", deviceProp.maxThreadsPerBlock);

        // Maximum sizes of each dimension of a block
        printf("  Max sizes of each dimension of a block:        %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);

        // Maximum sizes of each dimension of a grid
        printf("  Max sizes of each dimension of a grid:         %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        //
        printf("warp size                                        %d\n", deviceProp.warpSize);
        printf("regs per multiprocessor                          %d\n", deviceProp.regsPerMultiprocessor);
        printf("regs per block                                   %d\n", deviceProp.regsPerBlock);
        printf("clock                                            %d\n", deviceProp.clockRate);
        printf("memory clock                                     %d\n", deviceProp.memoryClockRate);
        printf("shared mem per block                             %d\n", deviceProp.sharedMemPerBlock);
        printf("unifiedAddressing                                %d\n", deviceProp.unifiedAddressing);
        printf("unifiedFuncPointers                              %d\n", deviceProp.unifiedFunctionPointers);

        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  Number of multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  CUDA Cores per multiprocessor: " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Max threads per multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
    }

    return 0;
}
