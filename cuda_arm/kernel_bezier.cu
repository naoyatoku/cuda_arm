﻿#include "commonmodule.h"
#include "arm.h"
#include "bezier.h"
#include "kernels.h"

//#define MAX_PATH    4096    //パス最大値
//- rdc = true

//テンプレートを使用可能にする
//template bool _kernel<bezier>(arm* _arm, void* _p_path_class, float tgt_spd, int dir, _cood* tgt_pos, arm* _path, int* _path_idx, int fin_condition);

/*
void bezier_path()
{
	cpu_gpu_mem<bezier> _bezier;                       //
    //※デフォルトコンストラクタでアサートしてしまっている。
	cpu_gpu_mem<arm>	_arm;                          //アーム
    cpu_gpu_mem<arm>    _path(MAX_PATH);               //アームの状態を直接記録します。各リンクの動作や速度もトレースできるので
    cpu_gpu_mem<int>    _path_idx;                     //パスの現在インデックスです。+		gpu	0x0000000705804400 {???}	int *
    cpu_gpu_mem<_cood>  _tgt_pos(2);                   //始点～終点の位置


    printf("line path start!\r\n");
    //==========================================================================
    //  軌跡の設定
    //==========================================================================
    _tgt_pos(CPU,START) = _cood(vec2(150,250),PI/2);   //始点
    _tgt_pos(CPU,END) = _cood(vec2(230,330),PI/2);      //終点
    const float tgt_spd = 30.0;                             //10mm/sec という意味で設定しますがd2096
    //ベジェ曲線の設定
    _bezier(CPU,0).set_params( _tgt_pos(CPU,START) , vec2(180,300), vec2(210,170), _tgt_pos(CPU,END) , 1000);
    //==========================================================================
    //  軌跡記録の初期化
    //=========================================================================
    {   //path
        memset((void*)&_path(CPU) , 0 , _path.size);        //
        _path_idx(CPU) = 0;
    }
    //gpuへ転送
    {
        _bezier.Transfer_to_GPU();
        _path.Transfer_to_GPU();
        _path_idx.Transfer_to_GPU();
        _tgt_pos.Transfer_to_GPU();
        _bezier.Transfer_to_GPU();
    }
    //==========================================================================================================
    //==========================================================================================================
    {
        //カーネルからカーネルが呼べないので
        {
            //まず停止予定位置から、マイナス方向に目標速度になるまで動作させる。
            //停止する開始時間を記録するようにします。
            int approx_dec_time;   //
            {
                _arm(CPU).set(_tgt_pos(CPU, END));  _arm.Transfer_to_GPU();     //アームを終点位置へ
                //終点→目標速度に到達するまでの軌跡計算です。
                   _kernel<bezier>(_arm.gpu ,  _bezier.gpu, tgt_spd, 0 , &_tgt_pos.gpu[END], _path.gpu, _path_idx.gpu, FIN_SPD);      //
//bool _kernel(arm * _arm , void  *_p_path_class , float tgt_spd, int dir , _cood *tgt_pos,arm *_path , int *_path_idx , int fin_condition);
                approx_dec_time = *_path_idx.Transfer_to_CPU();
                _path_idx(CPU) = 0; _path_idx.Transfer_to_GPU();        //0にして戻します。
                //debug dump
            }
        }
    }

}

*/