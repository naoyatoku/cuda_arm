﻿#include "commonmodule.h"
#include "arm.h"
#include "bezier.h"
#include "kernels.h"

#define MAX_N_PATH    8192    //パス最大値


void bezier_path()
{
	cpu_gpu_mem<bezier> _bezier;                       //
    //※デフォルトコンストラクタでアサートしてしまっている。
	cpu_gpu_mem<arm>	_arm;                          //アーム
    cpu_gpu_mem<arm>    _path(MAX_N_PATH);               //アームの状態を直接記録します。各リンクの動作や速度もトレースできるので
    cpu_gpu_mem<int>    _path_idx;                     //パスの現在インデックスです。+		gpu	0x0000000705804400 {???}	int *
    cpu_gpu_mem<_cood>  _tgt_pos(2);                   //始点～終点の位置


    printf("line path start!\r\n");
    //==========================================================================
    //  軌跡の設定
    //==========================================================================
    _tgt_pos(CPU,START) = _cood(vec2(150,250),PI/2);   //始点
    _tgt_pos(CPU,END) = _cood(vec2(230,330),PI/2);      //終点
    const float tgt_spd = 30.0;                             //10mm/sec という意味で設定しますがd2096
    //ベジェ曲線の設定
    _bezier(CPU,0).set_params( _tgt_pos(CPU,START) , vec2(180,300), vec2(210,170), _tgt_pos(CPU,END) , 1000);
    _tgt_pos(CPU,START).write_add_info<bezier_pos>( _bezier(CPU).pos(0.0) );    //始点と終点にベジェの始点終点情報を加えます。※ちょっと面倒
    _tgt_pos(CPU,END).write_add_info<bezier_pos>( _bezier(CPU).pos(1.0) );

    //==========================================================================
    //  軌跡記録の初期化
    //=========================================================================
    {   //path
        memset((void*)&_path(CPU) , 0 , _path.size);        //
        _path_idx(CPU) = 0;
    }
    //gpuへ転送
    {
        _bezier.Transfer_to_GPU();
        _path.Transfer_to_GPU();
        _path_idx.Transfer_to_GPU();
        _tgt_pos.Transfer_to_GPU();
    }
    //==========================================================================================================
    //==========================================================================================================
    {
        //=====================================================================
        //      終点～逆に移動していき、目標速度にたっするまでの時間を記録する。
        //=====================================================================
        int approx_dec_time;   //
        {
            {   //アーム初期状態を記録
                _arm(CPU).set(_tgt_pos(CPU, END));
                _arm(CPU).write_add_info<bezier_pos>(
                    bezier_pos( 1.0 , _bezier(CPU).find_len_from_t(CPU,1.0) )
                );    //終点の位置情報を記録します。
            }
            _arm.Transfer_to_GPU();     //アームを終点位置へ
            //終点→目標速度に到達するまでの軌跡計算です。
            if(_kernel<bezier>(_arm.gpu ,  _bezier.gpu, tgt_spd, DIR_M , &_tgt_pos.gpu[END], _path.gpu, _path_idx.gpu, _path.size, FIN_SPD) != KERNEL_FIN_OK) {
                printf("[phase 1]error:kernel error\r\n");
                return;
            }
//bool _kernel(arm * _arm , void  *_p_path_class , float tgt_spd, int dir , _cood *tgt_pos,arm *_path , int *_path_idx , int fin_condition);
            approx_dec_time = *_path_idx.Transfer_to_CPU();
            _path_idx(CPU) = 0; _path_idx.Transfer_to_GPU();        //0にして戻します。
        }
        //=====================================================================
        //      開始位置から終点位置まで動作させてしまう。
        //=====================================================================
        {
            {   //アーム初期状態を設定
                _arm(CPU).set(_tgt_pos(CPU, START));
                _arm(CPU).write_add_info<bezier_pos>(
                    bezier_pos( 0.0 , _bezier(CPU).find_len_from_t(CPU,0.0) )       //付加情報
                );
                _arm.Transfer_to_GPU();                                             //GPUへ転送
            }
            //bezierの場合は、
            if(_kernel<bezier>(_arm.gpu, _bezier.gpu, tgt_spd, DIR_P, &_tgt_pos.gpu[END] , _path.gpu, _path_idx.gpu,_path.size , FIN_POS_PASS)!=KERNEL_FIN_OK) {
                //この場合は、tが1.0の頭打ちになった場合があるので、許容するかどうかをここで決めます。
                _arm.Transfer_to_CPU(); //現在のARMを取得して、目的位置に近ければOK
                if( (1.0 - _arm(CPU).read_add_info<bezier_pos>().t) < 0.01) {    //ある程度近い場合にはOKとする。これは99%です。
					printf("OK\r\n");
				}
				else {
                    printf("[phase 2]error:kernel error\r\n");
                    return;
                }
			}
        }
        //=====================================================================
        //    停止開始位置を探します。
        //=====================================================================
        {
#if 1       //armがどこまですすんだか確かめます。

            _arm.Transfer_to_CPU();
                //
#endif
            const int n_try=approx_dec_time + (approx_dec_time/5);        // ± 推定減速開始時刻の半分 + 10%程度の範囲を探す。
            //とりあえず、全部ではなくていちいち上書きするようにする。
            const int t_max=256;        //これは少なくとも、停止時間(400ms分ないといけないと思う。)
            //一度軌跡を戻します。
            _path.Transfer_to_CPU();        //時間がかかるので必要な部分だけ取り出したいが
            _path_idx.Transfer_to_CPU(); 

            //approprox_dec_timeを目安に、目標位置に一番近い場所で止まるタイミングを探します。(×n_try回数分)
            //※各パスを記録して検証する。
            cpu_gpu_mem<arm>decpath(t_max);
            cpu_gpu_mem<int>decpath_idx;
            cpu_gpu_mem<arm>_temparm;

            //一番近くに停止する位置を決めていきます。
            _cood best_arm;
            float min_dist= 999999.9;    //有効数字7桁

            //最小パスを保存しておくパスバッファをを作ります。
            //異常判定のために変な値を初期設定しておく。
            arm* best_path = new arm[t_max];
            int best_path_idx;
            int best_timing=-1;
            for (int t = 0; t  < n_try ; ++t) {
                int time = (_path_idx(CPU) - 1 - approx_dec_time) - (t - (n_try / 2));  //今回評価する時間です。
                {
                    decpath_idx(CPU) = 0;
                    decpath_idx.Transfer_to_GPU();                    
                    _temparm(CPU) = _path(CPU,time );    //最終位置からtさかのぼって考える(_path_idx - 1が、最終インデックスです。
                    _temparm.Transfer_to_GPU();                             //gpuに送る
                }
                printf("dec try t=[%d]\r\n", t);
                if (_kernel<bezier>(_temparm.gpu, _bezier.gpu, 0, DIR_P, &_tgt_pos.gpu[END], decpath.gpu, decpath_idx.gpu, decpath.size, FIN_SPD) != KERNEL_FIN_OK) {
                    printf("kernel error\r\n");
                }else{
                    //最小距離を判定していきます。
                    _temparm.Transfer_to_CPU();     //アームが最終的に到達した位置です。
                    //最小を更新していきます。初登録の場合は強制的にベストとします。
                    if ( _tgt_pos(CPU,END).distance(_temparm(CPU)) < min_dist) {
                        min_dist = _tgt_pos(CPU,END).distance(_temparm(CPU));    //最小距離を更新委
                        best_arm = _temparm(CPU);                       //アーム最終位置を更新
                        best_timing = time;                             //減速開始時刻です。
                        //ベストパスをコピーします。
                        {
                            decpath.Transfer_to_CPU();
                            decpath_idx.Transfer_to_CPU();
                            best_path_idx = decpath_idx(CPU);
                            memcpy((void*)&best_path[0], (const void*)decpath.cpu, best_path_idx * sizeof(arm)); // 必要な分だけコピー
                        }
                    }
                }
                _dump_path(decpath, decpath_idx);
            }
            //このループが終わった時点でbest_pathとbest_path_idxとbest_timingに減速開始の情報が保存されています。
            _Assert(best_timing >= 0, "best_timing is not found");
            //大本のpathに減速を連結していきます。
            {
                _Assert(best_timing + best_path_idx < _path.size , "path overflow (renketu)");
                memcpy((void*)(_path.cpu + best_timing), (const void*)best_path, best_path_idx * sizeof(arm));   //NGならforでoperator=を使っていく。
                _path_idx(CPU) = best_timing + best_path_idx;
                _path.Transfer_to_GPU();
                _path_idx.Transfer_to_GPU();

                for (;;) {
                    _draw_path(_path, _path_idx, 2);
                }
            }
        }
    }
    return;
}
