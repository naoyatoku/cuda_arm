#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "jacobi.h"


static	double	_l1=290.0;
static	double	_l2=290.0; 
static	double	_l3=135.0;

//このモジュールで使う間接のアーム長さを登録出来るようにしておきます。
void set_jacobi_parameter(double l1,double l2 , double l3)
{	
	_l1=l1;
	_l2=l2;
	_l3=l3;
}
//
Matrix3d jacobian(double th1, double th2, double th3)
{
	Eigen::Matrix3d J;

	J <<	-_l1*sin(th1)	-	_l2 * sin(th1 + th2) - _l3 * sin(th1 + th2 + th3)	, 	-_l2*sin(th1 + th2) - _l3 * sin(th1 + th2 + th3)	, -_l3 * sin(th1 + th2 + th3),
			_l1*cos(th1)	+	_l2 * cos(th1 + th2) + _l3 * cos(th1 + th2 + th3)	,	_l2*cos(th1 + th2) + _l3 * cos(th1 + th2 + th3)	,	_l3*cos(th1 + th2 + th3),
			1, 1, 1;
	return J;
}

//
bool inv_jacobi(double th1,double th2 ,double th3 , Matrix3d &J_inv) 
{
	// Calculate the Jacobian
	Matrix3d J = jacobian(th1, th2, th3);

	// Check if the determinant is non-zero
	{
		double det = J.determinant();
		if (std::abs(det) < 1e-6) {
			std::cerr << "The determinant is zero. The inverse Jacobian does not exist." << std::endl;
			return false;
		}
	}

	// Calculate the inverse Jacobian
	J_inv = J.inverse();

	// Print the Jacobian and its inverse
	std::cout << "Jacobian:\n" << J << std::endl;
	std::cout << "Inverse Jacobian:\n" << J_inv << std::endl;

	return true;
}

