#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "inv_kinetic.h"
#include "linear.h"
#include "arm.h"
#include "CommonModule.h"
#include "draw.h"
#include "kernels.h"

//やはり1ブロックあたりのスレッド数に限りがある(1024まで)なので、ブロックに分けていかないといけない。
//RTX4060Laptopのデバイスでは

//SM 24個
//SM 1つあたり 1536(512*3)スレッド  →   ×24 => 36864スレッド平行にできる。
//1ブロック当たり512スレッドとすると、72ブロックが並列処理の最大数。

//(共有メモリ)
//SM 1つあたり 64kbyteが限界。
//ブロックあたり    48byteの制約がある。
//SMあたり実行するように想定したブロック数（今回の場合は3個）分、合計して64kbyteを超える場合、平行に処理されなくなるので、注意する。
//  (実行はできてしまうが、順番になってしまう可能性がある。)
//  ※共有メモリはSM毎に

//（レジスタ）
//1スレッドあたりの最大レジスタ数： 255
//1 SMあたりのレジスタ数        ：  65,536
//共有メモリとは関係ない。

//(グローバルメモリ)
//RTX4060Laptopの場合、  8Gbyteある。


//方向を示す定義


//カーネルの結果をチェックする。
__host__
bool check_kernl_error()
{
    hipError_t s;
    s = hipGetLastError();
    if (s != hipSuccess) {
        printf("kernel failed: %s\n", hipGetErrorString(s));
        return false;
    } //
    return true;
}
//一貫性を持たせるため（カーネル分割するので）関数状態にします。
//dirは、速度に符号をつけて一緒にしてもよい。
__device__ inline
_cood _tgt_cood(const arm* _arm , const linear*_line ,float tgt_spd , int dir,int idx,int n_all_thread)
{
    float d_spd  = tgt_spd - _arm->d.spd;                             //目標速度にむけての加速度です。(mm/sec)

//  float dx =  dir * (spd_span / n_all_thread) * (idx + ( (tgt_spd > _arm->d.spd) ? 1 : 0)) / 1000; //1ms分なので、 spd rad/sec -> rad/msec に修正        idx=511で bloclDim.xとなるように

    //-----------------------------------------------------------------------------------------------------------------
    //  ※toku 
    //  ここは、現在速度～目標速度の間だけだと、減速が効かないので、0もしくは、あるていどの減速までを範囲に含める必要がある。
    //-----------------------------------------------------------------------------------------------------------------

    //　(現在速度＋目標速度への加速度)/1000 (1msあたりに換算)
    float dx = ((d_spd / n_all_thread) * idx)  / 1000; //
    //デバッグしやすいように計算を分けます。
    dx += _arm->d.spd / 1000;   //現在速度で1mあたりに進む距離
    //dxこれが現在速度からの加速分です。現在速度の

    _cood c(_line->progress(*_arm,dx,dir),_arm->rad);   //デバッグ用に変数にします。
    return c;
}
//1msおきで進める位置を求める。
__global__
void line_calc_kernel(arm* _arm , linear* _line ,int* block_result , float tgt_spd , int dir)
{
    //今回のケースの場合、SMあたり3ブロック実行させたいので、21kByteまでに抑える必要がある。
    //リダクション（高速実行するために）
    __shared__ int _reduction[N_THREAD];        //int*512 -> 2kbyte
//    bool acc = (tgt_spd > _arm->d.spd ) ? true : false;       //加減速スイッチです。

    //ブロックとスレッドの組み合わせで成功失敗の
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;                          //
    link_stat s = _arm->move_able(_tgt_cood(_arm , _line, tgt_spd, dir,idx,N_ALL_THREAD ));      //

    //リダクション用のバッファに結果を入れて言います。
//    _reduction[threadIdx.x] = s.stat == MOV_OK ? idx : (acc?-1:N_ALL_THREAD) ;     //成功したら自分のインデックスを入れる失敗したら
     //_reductionにはいる数値（インデックス）は、大きいほうがもっとも理想に近いものになっています（加速、減速時ともに）
    _reduction[threadIdx.x] = s.stat == MOV_OK ? idx : -1 ;     //成功したら自分のインデックスを入れる失敗したら
    __syncthreads();        //あるブロック内部のスレッドの同期です。
    //==================================================================================
    // リダクションでブロック単位の評価を行います。
    //===================================================================================
    _do_reduction(_reduction,blockDim.x);

    //ここでブロック内のチャンピオンデータが_reduction[0]に保管されている。自分のブロックデータを更新します。
    if(threadIdx.x==0){
        block_result[blockIdx.x] = _reduction[0];
#if 0   //
//        if (blockIdx.x < 3) {
//            printf("\x1B[%d;%dH ac[%d]bl[%d]r[%d]\t",0, blockIdx.x*100,  acc, blockIdx.x, _reduction[0]);
            printf("bl[%d]a[%d]r[%d]\t", blockIdx.x,acc, _reduction[0]);
//        }
#endif
    }
}
//一番ベストな位置へアームを動作する
__global__ void line_move_kernel(arm* _arm , linear* _line ,int* block_result , float tgt_spd ,_cood *tgt_pos ,  int dir, bool* fin,arm* _path, int* _path_idx , int fin_condition)
{
    //まずブロック毎のデータの中でさらに一番いいものを選択する。
    _do_reduction(block_result , blockDim.x );

    //いちいちCPUに戻さなくてもいいように、アームをここで動作させ、終了判定をします。
    if(threadIdx.x==0){ //
        _gpuAssert(block_result[0] >= 0, "move_kernel():no available result\r\n");       //有効なインデックスがなかった場合はエラーです。
        _cood tgt =_tgt_cood( _arm , _line , tgt_spd,dir,block_result[0],N_ALL_THREAD);
        _arm->move(tgt);                                   //動かす。

#if 0
        printf("\r\n ====> [%d]ms(%f,%f),spd(%f) acc(%f) block_result[%d]\r\n", *_path_idx , _arm->x ,_arm->y , _arm->d.spd , _arm->d.acc, block_result[0]);
#endif

//        printf("[%d]t:%lf,spd:%lf,acc%lf,dlt:%lf\n", *n_path, _arm->rad, _arm->wd.spd, _arm->wd.acc, fabs(_arm->wd.spd) - spd);
        _path[(*_path_idx)++] = *_arm;    //アーム記録してもいいかも

        if (*_path_idx > MAX_PATH-1) {  //overflow
            printf("\r\npath overflow\r\n");
            *fin = true;
            goto _fin;
        }

        //終了判定します。
        {
            *fin = false;
            switch(fin_condition){
                case FIN_SPD:
                    if (_equal<float>(_arm->d.spd, tgt_spd,0.1) == true) {
                        *fin = true;
                    }
                    break;
                case  FIN_POS_PASS:
                    if(dir == DIR_P){   //プラス方向に進行している場合、
                        if( _arm->x >= tgt_pos->x ){
                            *fin=true;
                        }
                    }else{
                        if( _arm->x <= tgt_pos->x ){
                            *fin=true;
                        }
                    }
            }
        }
    }
_fin:
    __syncthreads();    //これいらないか？
#if 0   //break用
    if (threadIdx.x == 1) {
        if (*fin == true) {
            //ダンプしてみる。
            for (int i = 0; i < *_path_idx ; ++i) {
                const arm& cur = _path[i];
                printf("[%d](%f,%f,%f)spd(%f)\r\n", i, cur.x, cur.y, cur.rad, cur.d.spd);
            }
        }
    }
    __syncthreads();    //これいらないか？
#endif
    //
}
//もう一個、目的位置まで動作するカーネルを作る。

//__global__
//toku 最初、ここはカーネル呼び出しのはずだったが、うまくいかなかったのでホストコードにした。
//現状、ポインタはGPUアドレスを受けていますがこのままにしておきます。のちにカーネルにできるならしたいので
__host__
bool line_kernel(arm * _arm , linear * _line , float tgt_spd, int dir , _cood *tgt_pos,arm *_path , int *_path_idx , int fin_condition)
{
    const dim3 threads( N_THREAD);
    const dim3 blocks(N_BLOCK);
    cpu_gpu_mem<int>	block_result(N_BLOCK);		//これが全部の結果を保持するようにします。一度の
    cpu_gpu_mem<bool>fin;

    esc_clr();  //画面はクリアする。


    //※アーム状態はカーネル呼び出し前に設定してください。
    for (fin(CPU) = false;  fin(CPU) != true; fin.Transfer_to_CPU() ) {      //この処理に時間がかかるのでこのループ自体も
        line_calc_kernel << <blocks, threads >> > ( _arm , _line , block_result.gpu , tgt_spd, dir);
        hipDeviceSynchronize();
        if (check_kernl_error() != true) {
            goto _error;
        }
        //この時点でベストが見つかっているとする。
        line_move_kernel << <1, blocks >> > (_arm , _line , block_result.gpu , tgt_spd, tgt_pos , dir , fin.gpu , _path , _path_idx ,fin_condition );
        hipDeviceSynchronize();
        if (check_kernl_error() != true) {
            goto _error;
        }
    }

    return true;
_error:
    return false;
}

//void line_calc_kernel(arm* _arm , linear* _line ,int* block_result , float tgt_spd , int dir)
//__global__ void line_move_kernel(arm* _arm , linear* _line ,int* block_result , float tgt_spd ,int dir, bool* fin,_cood* _path, int* _path_idx )



//=======================================================================================================================================================================================
//		lineパス計算（カーネル呼び出し）
//=======================================================================================================================================================================================

void _dump_path(cpu_gpu_mem<arm>&path , cpu_gpu_mem<int>&path_idx,bool disp_link/*=false*/,int wait/*=0*/)
{
    path.Transfer_to_CPU();
    path_idx.Transfer_to_CPU();
    for (int i = 0; i < path_idx(CPU); ++i) {
        const arm& cur = path(CPU, i);
        //描画できるように
        _arm = cur;
        printf("[%d](%f,%f,%f)spd(%f)\r\n", i, cur.x, cur.y, cur.rad, cur.d.spd);
        if(disp_link){
        for (int l = 0; l< 3; ++l){
            printf("        L%d(pos[%f(deg:%f)]spd[%f]\r\n", l, cur.lnk(l).r ,RAD2DEG( cur.lnk(l).r) ,  cur.lnk(l).d.spd);
        }
        }
//        printf("\r\n");
       // イベントをシグナル状態に設定
        SetEvent(_draw_req_event);

        ::Sleep(wait);
    }
}



static vec2 _v; //debug
//スレッドのx,yで、同じブロックで行ってみる。
void line_path()
{

	//なんかあれなんでここで
	cpu_gpu_mem<linear> _line;                           //
	cpu_gpu_mem<arm>	_arm;                            //アーム
    cpu_gpu_mem<arm>    _path(MAX_PATH);                    //アームの状態を直接記録します。各リンクの動作や速度もトレースできるので
    cpu_gpu_mem<int>    _path_idx;                          //パスの現在インデックスです。+		gpu	0x0000000705804400 {???}	int *
    cpu_gpu_mem<_cood>  _tgt_pos(2);                   //始点～終点の位置

    printf("line path start!\r\n");
    //==========================================================================
    //  軌跡の設定
    //==========================================================================
    _tgt_pos(CPU,0) = _cood(vec2(150,250),PI/2);   //始点
    _tgt_pos(CPU,1) = _cood(vec2(230,330),PI/2);   //終点

#if 0       //始点と終点の位置を確認します。
    {
        _arm(CPU).set(_tgt_pos(CPU, START));
            _v = _arm(CPU).lnk(0).linked_vect();
            _v = _arm(CPU).lnk(1).linked_vect();
            _v = _arm(CPU).lnk(2).linked_vect();
        _arm(CPU).set(_tgt_pos(CPU, END));
             _v = _arm(CPU).lnk(0).linked_vect();
            _v = _arm(CPU).lnk(1).linked_vect();
            _v = _arm(CPU).lnk(2).linked_vect();
    }
#endif
    const float tgt_spd = 30.0;                             //10mm/sec という意味で設定しますがd2096
    _line(CPU) = linear( 1.0 , 100.0 );                     //直線の設定

    //==========================================================================
    //  もろもろ初期設定です。
    //
    //  軌跡となる直線を設定する。(開始時のアーム位置が(100,200)として、
    //  傾き1.0の直線→切片は100
    //=========================================================================
    {   //path
        memset((void*)&_path(CPU) , 0 , _path.size);        //
        _path_idx(CPU) = 0;
    }
    //gpuへ転送
    {
        _line.Transfer_to_GPU();
        _path.Transfer_to_GPU();
        _path_idx.Transfer_to_GPU();
        _tgt_pos.Transfer_to_GPU();
    }
    //==========================================================================================================
    //==========================================================================================================
    {
        //カーネルからカーネルが呼べないので
        {
            //まず停止予定位置から、マイナス方向に目標速度になるまで動作させる。
            //停止する開始時間を記録するようにします。
            int approx_dec_time;   //
            {
/*                cpu_gpu_mem<_cood> dec_start;           dec_start(CPU) = *_arm.Transfer_to_CPU();   //さきほど計算した_armの位置（減速開始位置）を記録する。
                dec_start.Transfer_to_GPU();                                                        //GPUへ転送
*/
                _arm(CPU).set(_tgt_pos(CPU, END));  _arm.Transfer_to_GPU();
                line_kernel(_arm.gpu ,  _line.gpu, tgt_spd, DIR_M, &_tgt_pos.gpu[END], _path.gpu, _path_idx.gpu, FIN_SPD);      //
                approx_dec_time = *_path_idx.Transfer_to_CPU();
                _path_idx(CPU) = 0; _path_idx.Transfer_to_GPU();        //0にして戻します。
                //debug dump
#if 0
                {
                    printf("idx=%d , pos=(%f,%f)\r\n" , approx_dec_time , );
                }
#endif
            }

            //ここで、現在の_arm位置が、減速を開始するおおよその位置です。これを記録します。

            //アームを開始位置にリセットし減速開始位置まで動作させる。
            {
                _arm(CPU).set(_tgt_pos(CPU, START));
                _arm.Transfer_to_GPU();                         //GPUへ転送
                //やはりここで後の
                line_kernel(_arm.gpu, _line.gpu, tgt_spd, DIR_P, &_tgt_pos.gpu[END] , _path.gpu, _path_idx.gpu, FIN_POS_PASS);      //減速開始位置に
            }


//            line_kernel()」//これを何種類かやって、ぴったりにとまるところを探す。
            {
                const int n_try=5;
                const int t_max=256;
                //一度軌跡を戻します。
                _path.Transfer_to_CPU();        //時間がかかるので必要な部分だけ取り出したいが
                _path_idx.Transfer_to_CPU(); 
                //100個くらいできるか?
                cpu_gpu_mem<arm>decpath[n_try] = { cpu_gpu_mem<arm>(t_max) ,cpu_gpu_mem<arm>(t_max)  , cpu_gpu_mem<arm>(t_max) , cpu_gpu_mem<arm>(t_max) ,cpu_gpu_mem<arm>(t_max) };       //
                cpu_gpu_mem<int>decpath_idx [n_try];        //
                cpu_gpu_mem<arm>_temparm;

                for (int t = 0; t  < n_try ; ++t) {
                    _temparm(CPU) = _path(CPU,_path_idx(CPU) - 1  - t) ;    //最終位置からtさかのぼって考える(_path_idx - 1が、最終インデックスです。
                    _temparm.Transfer_to_GPU();                         //gpuに送る
                    printf("dec try [-%d]\r\n", t);
                    line_kernel( _temparm.gpu , _line.gpu, 0, DIR_P, &_tgt_pos.gpu[END], decpath[t].gpu, decpath_idx[t].gpu, FIN_SPD);
                    _dump_path(decpath[t], decpath_idx[t]);
                    //本来、ここで一番いいものを選定します。
                    //
                    //      ※まだ作っていません
                    //
                }

                //toku とりあえず最後のを採用して、pathと結合する。
                {
                    const int t=n_try-1;
                    //
                    decpath[t].Transfer_to_CPU();
                    decpath_idx[t].Transfer_to_CPU();
                    //
                    const int path_renketsu_point = _path_idx(CPU) - 1  - t;//ここから連結します。
                    for(int i =0 ; i < decpath_idx[t](CPU) ; ++i ){
                        _path(CPU,path_renketsu_point + i ) = decpath[t](CPU,i);
                    }
                    //終了位置を合わせます。
                    _path_idx(CPU) = path_renketsu_point + decpath_idx[t](CPU);

                    //GPUも合わせておきます・・・ヘんか？あとでGPUを使われてもいいように
                    _path.Transfer_to_GPU();
                    _path_idx.Transfer_to_GPU();
                }
            }
        }
    }
    for (int i=0;i<1;++i) {
        _dump_path(_path, _path_idx, true);
//    ::Sleep(1000);
    }

    //各リンクの軌跡を記録します。これは関数にするか
    {
        csv bin[3] = { csv("l1.bin",true) , csv("12.bin",true) , csv("l3.bin",true) };  //こちらは軸指示用(bin)
        csv log("arm.log",true);
        for(int i=0 ; i<_path_idx(CPU) ; ++i) {     //
            arm & _arm = _path(CPU,i);  //この瞬間のアーム状態です
            log.writef("%d,%f,%f,%f,%f,%f," , i,_arm.x,_arm.y,_arm.rad,_arm.d.spd,_arm.d.acc);  //アームの状態。
            for(int l=0;l<3;++l){
                bin[l].bin_write<float>(_arm.lnk(l).r);
                log.writef("%f,%f,%f," , _arm.lnk(l).r, _arm.lnk(l).d.spd,_arm.lnk(l).d.acc);   //リンクの Θ,spd,accを記録
            }
            log.write("\r\n");
        }
    }
    //

_error:;
    printf("end\r\n");

}

