#include "hip/hip_runtime.h"
﻿#include "link.h"
#include "units.h"
#include <math.h>

#if 0
__device__ __host__ double link::nearest(const double new_rad){
	double min =2*PI;int min_n=0;
	for(int n = -1 ; n < 2 ; ++n ){
		double d;
		if( (d = fabs(r - (new_rad + (2*PI*n)) )) < min ){	//最小値を見つけていきます。
			min = d;
			min_n=n;
		}
	}
	//見つけた最小のものにnewposを変更します。
	double _new_rad = new_rad + (2.0*PI*min_n);
	return _new_rad;
}

//位置の指示ができるかを判断します。
/*bool link::judge_set(double newpos) {
	//まず、位置の差分が自分の加減速速度より小さくなければなりません。
	_cood _newpos( vec2(_l*cos(newpos),_l*sin(newpos)) , newpos);	//これが新しい式です。	
	_cood me = *this;	//自分のコピーです。
	me = _newpos;		//セットするとで
  
	//加速も減速もせずに到達する位置
	double next_pos = _pos + (_spd / 1000);  //現在の位置＋現在速度です。(1msの場合、1000で割る必要がある)
	//要求された位置との差が、差第加速度を超えなければ動ける
	//角度が近い場所を探します。どうやってさがすんか
	double near_newpos = nearest_pos(next_pos, newpos);
	if (fabs(next_pos - nearest_pos(next_pos,newpos)) <= _max_acc) {       //大きくなるにせよ小さくなるにせよ、最大加速を超えなければＯＫ
		return true;
	}
	return false;
}
*/
__device__ __host__ int	link::move_able(const double new_rad, double ms)
{
//	double	_new_rad = nearest(new_rad);
//	_cood _newpos( vec2(_l*cos(_new_rad),_l*sin(_new_rad)) , _new_rad);	//これが新しい角度の_coodです。
	//自分のコピーを動かしてみて、その加速度や速度を調べます。
	link me(*this);	
	me.move(new_rad, false);	//自分のコピーを動かしてみます。
	//
	//これ、加速度がある程度誤差をもってもいいようにするか？
	//加速度が小さい場合、

	//ここでは、加速がマイナス側に大きすぎたのか、プラス側に大きすぎたかだけを返すようにします。
	if( fabs(me.d.acc) - _max_acc > (_max_acc/10) ){		//加速（減速)が大きすぎた
									//^^^^^^^^^^これなんだっけ? (_max_acc +　10％　は許容するようにしている？)

		//多き過ぎか小さすぎかを返します
		//現在の速度と、結果加速度が同じ方向かどうかの判定です。

		//アーム動作の補正の方向を見つけます。
		//これは次回どちらの方向に向かえばいいかの方向をあわらします。
		//アームの動きに対して、各リンクは非線形に動作する。
		//この瞬間の勾配の値をもとにしないといけない。

		if(me.d.acc < 0.0 ){	return MOV_DEC_OVER;	}	//加速度がマイナス側に大きすぎた
		return MOV_ACC_OVER;								//加速度がプラス側に大きすぎた
/*
		if( (this->d.spd) * me.d.acc  >= 0.0 ) {		//加速度と現在の速度の符号が同じならば、加速がおおきすぎると答えます。
			return MOV_ACC_OVER;		
		}
		return MOV_DEC_OVER;		//反対方向へ動作させようとした場合には減速しきれないとします。
*/
	}
	if( fabs(me.d.spd) > _max_spd ){	//速度が大きすぎた場合
		return MOV_SPD_OVER;			//速度が大きすぎる。※いまのところ符号の判定が必要ない
	}
	return MOV_OK;
}
__device__ __host__ link &	link::move(const double new_rad, bool log,double ms) 
{
	double	_new_rad = nearest(new_rad);	//一番近い角度に変換sします。

	//debug いまの位置を計算します。
//	_cood _nowpos(vec2(_l*cos(rad), _l*sin(rad)), rad);
	rvec2 _newpos( l , new_rad );	//これが新しい式です。

	rvec2::move(_newpos,ms);
	//動いたときにログをとってみます。
	if (log) {
		//
		vec2 v(to_vec2());
		_log.writef("%lf,%lf,%lf,%lf,%lf\n", v.x, v.y, r, d.spd, d.acc);
	}
	return *this;
}

//基準軸からの角度を返します。
/*double link::total_t (void)const {
	double t = _pos;		//自分自身の位置
	if(_parent){
		t+=_parent->total_t();
	}
	return t;
}*/
//この関節の座標を返します（自分が接続しているリンクの座標を考慮した、自分自身の位置を返します。
//自分自身は、自分の軸が原点にあるときの
//
//double link::total_angle(void)const;
__device__ __host__ double	link::total_angle(void)const
{
	double th=0;
	if (_parent)	//親がいる場合には親までの全関節の角度です。
	{
		th = _parent->total_angle();
	}
	return th + r;		//自分自身の
}

__device__ __host__ vec2 link::linked_vect(void) const {
	vec2 v(rvec2(l, total_angle()));	//これが自分自身のベクトル。で(ややこしいですが極形式で登録して直交座標に変換してます。)
	//親がいる場合には親の先端座標を足します。
	if (_parent)
	{
		v += _parent->linked_vect();	//親までの先端座標を足します。（ここが、自分自身の根元の座標になる)
	}
	return v;		//座標は
}
//


#endif